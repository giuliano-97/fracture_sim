#include "hip/hip_runtime.h"
#include "CudaSupport.h"
#include "stdio.h"


// GPU Error check helper
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU ASSERT: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace CudaSupport {

const unsigned int MaxBlockSize = 512;

unsigned int blockSize;
unsigned int numBlocks;

// Device constants - initialized once for usage
__device__ __constant__ unsigned int numOfParticles;
__device__ __constant__ unsigned int hashBinsMaxSize;
__device__ __constant__ unsigned int hashBinsNum;
__device__ __constant__ unsigned int hashBinsNumHalf;
__device__ __constant__ double gridCellSize;
__device__ __constant__ unsigned int p1 = 73856093;
__device__ __constant__ unsigned int p2 = 19349663;
__device__ __constant__ unsigned int p3 = 83492791;
__device__ __constant__ double dt;
__device__ __constant__ double half_dt;
__device__ __constant__ double Kc;
__device__ __constant__ double3 gravity;
__device__ __constant__ double stiffness;
__device__ __constant__ double3 x_offset;
__device__ __constant__ double3 y_offset;
__device__ __constant__ double3 z_offset;
__device__ __constant__ double collisionThreshold;
__device__ __constant__ double lambda;



/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Start of device helpers of CUDA kernels //////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////
__device__ __forceinline__ double3 operator+(const double3& a, const double3& b)
{
	double3 out;
	out.x = a.x + b.x;
	out.y = a.y + b.y;
	out.z = a.z + b.z;
	return out;
}

__device__ __forceinline__ double3 operator-(const double3& a, const double3& b)
{
	double3 out;
	out.x = a.x - b.x;
	out.y = a.y - b.y;
	out.z = a.z - b.z;
	return out;
}

__device__ __forceinline__ double3 operator*(const double3& a, const double b)
{
	double3 out;
	out.x = a.x * b;
	out.y = a.y * b;
	out.z = a.z * b;
	return out;
}

__device__ __forceinline__ double3 operator*(const double b, const double3& a)
{
	double3 out;
	out.x = a.x * b;
	out.y = a.y * b;
	out.z = a.z * b;
	return out;
}

__device__ __forceinline__ double norm(const double3& v) {
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// End device helpers CUDA kernels //////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Start of CUDA kernels ////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////

__global__
void initializeKernel(
	unsigned int* hashCounts)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < hashBinsNum; i += stride) {
		hashCounts[i] = 0;
	}
}


__device__
unsigned int spatialHash(
	const double3& pos)
{
	long long int i = floor(pos.x / gridCellSize);
	long long int j = floor(pos.y / gridCellSize);
	long long int k = floor(pos.z / gridCellSize);

	return (((i * p1) ^ (j * p2) ^ (k ^ p3)) % hashBinsNumHalf) + hashBinsNumHalf;
}


__global__
void collectKernel(
	double3* positions,
	unsigned int* hashTable,
	unsigned int* hashCounts)
{
	// Fill the hash table
	unsigned int hash;
	unsigned int idx;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numOfParticles; i += stride) {
		hash = spatialHash(positions[i]);
		idx = hash * hashBinsMaxSize + atomicAdd(&hashCounts[hash], 1);
		hashTable[idx] = i;
	}
}


__global__
void detectCollisionsKernel(
	double3* positions,
	double3* forces,
	unsigned int* hashTable,
	unsigned int* hashCounts)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	// Collision detection
	for (int i = index; i < numOfParticles; i += stride) {
		// Get cells to check for each particle
		unsigned int cellsToCheck[8];
		unsigned int cellsToCheck_duplicates[8];
		double3 position = positions[i];

		// Hash all AABB vertices
		cellsToCheck_duplicates[0] = spatialHash(position - x_offset - y_offset - z_offset);
		cellsToCheck_duplicates[1] = spatialHash(position + x_offset - y_offset - z_offset);
		cellsToCheck_duplicates[2] = spatialHash(position - x_offset + y_offset - z_offset);
		cellsToCheck_duplicates[3] = spatialHash(position - x_offset - y_offset + z_offset);
		cellsToCheck_duplicates[4] = spatialHash(position + x_offset + y_offset - z_offset);
		cellsToCheck_duplicates[5] = spatialHash(position + x_offset - y_offset + z_offset);
		cellsToCheck_duplicates[6] = spatialHash(position - x_offset + y_offset + z_offset);
		cellsToCheck_duplicates[7] = spatialHash(position + x_offset + y_offset + z_offset);

		unsigned int numCellsToCheck = 0;

		bool dupl;
		for (int i = 0; i < 8; ++i) {
			dupl = false;
			for (int j = 0; j < numCellsToCheck; ++j) {
				if (cellsToCheck_duplicates[i] == cellsToCheck[j]) {
					dupl = true;
					break;
				}
			}
			if (!dupl) {
				cellsToCheck[numCellsToCheck++] = cellsToCheck_duplicates[i];
			}
		}
		
		// Check all the cells - if they are colliding, compute response
		unsigned int nextCell, start;
		for (int j = 0; j < numCellsToCheck; ++j) {
			nextCell = cellsToCheck[j];
			start = nextCell * hashBinsMaxSize;
			for (int k = start; k < start + hashCounts[nextCell]; ++k) {
				if (hashTable[k] != i) {
					double3 diff = positions[i] - positions[hashTable[k]];
					double distance = norm(diff);
					if (distance < 1e-9) continue;
					if (distance < collisionThreshold) {
						//printf("Particles %d and %d are colliding!\n", i, hashTable[k]);
						forces[i] =
							forces[i] + (Kc * pow(distance - collisionThreshold, 2) / distance) * diff;
					}
				}
			}
		}
	}
}


__global__
void advanceVelocitiesKernel(
	double3* velocities,
	double3* forces,
	double massInv)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numOfParticles; i += stride) {
		velocities[i] = velocities[i] + half_dt * massInv * forces[i];
	}
}


__global__
void advancePositionsKernel(
	double3* positions,
	double3* velocities)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numOfParticles; i += stride) {
		positions[i] = positions[i] + dt * velocities[i];
	}
}


__global__
void addBodyForcesKernel(
	double3* positions,
	double3* velocities,
	double3* forces,
	double particleMass)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numOfParticles; i += stride) {
		if (positions[i].y < 0) {
			// Restore position, otherwise particle might remain stuck under the floor (?)
			//positions[i].y = 0;
			// Reflect velocity
			velocities[i].y *= -1;
		}
		forces[i] = particleMass * gravity;
	}
}


__global__
void addSpringForcesKernel(
	double3* positions,
	double3* forces,
	int* adjs,
	unsigned int* adjsCounts,
	unsigned int* adjsStarts,
	double* restLengths,
	double* taus)
{
	int start, end;
	double epsilon, distance;
	double3 diff;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numOfParticles; i += stride) {
		start = adjsStarts[i];
		end = start + adjsCounts[i];
		for (int j = start; j < end; ++j) {
			if (adjs[j] != -1) {
				diff = positions[i] - positions[adjs[j]];
				distance = norm(diff);
				if (distance <= 1e-9) continue;
				epsilon = (distance / restLengths[j]) - 1;
				if (epsilon > taus[i]) {
					//printf("The spring between %d and %d broke!\n", i, adjs[j]);
					adjs[j] = -1;
					continue;
				}

				if (epsilon != 0) {
					forces[i] = forces[i] + diff * (-1 * stiffness * epsilon / distance);
				}
			}
		}
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// End of CUDA kernels //////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Start of simulation interface ////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////

void initializeSimulationParameters(
	unsigned int numOfParticles_host,
	unsigned int hashBinsNum_host,
	unsigned int hashBinsMaxSize_host,
	double gridCellSize_host,
	double dt_host,
	double Kc_host,
	double stiffness_host,
	double avgtau_host,
	double collisionThreshold_host,
	double lambda_host,
	thrust::device_vector<double3>& velocities,
	thrust::device_vector<double3>& forces,
	thrust::device_vector<double>& taus,
	thrust::device_vector<unsigned int>& hashTable,
	thrust::device_vector<unsigned int>& hashCounts)
{
	hipMemcpyToSymbol(HIP_SYMBOL(numOfParticles), &numOfParticles_host, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(hashBinsMaxSize), &hashBinsMaxSize_host, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(hashBinsNum), &hashBinsNum_host, sizeof(unsigned int));
	unsigned int hashBinsNumHalf_host = hashBinsNum_host / 2;
	hipMemcpyToSymbol(HIP_SYMBOL(hashBinsNumHalf), &hashBinsNumHalf_host, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(gridCellSize), &gridCellSize_host, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dt), &dt_host, sizeof(double));
	double half_dt_host = dt_host / 2;
	hipMemcpyToSymbol(HIP_SYMBOL(half_dt), &half_dt_host, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Kc), &Kc_host, sizeof(double));
	double3 gravity_host = make_double3(0, -9.81, 0);
	hipMemcpyToSymbol(HIP_SYMBOL(gravity), &gravity_host, sizeof(double3));
	hipMemcpyToSymbol(HIP_SYMBOL(stiffness), &stiffness_host, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(collisionThreshold), &collisionThreshold_host, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(lambda), &lambda_host, sizeof(double));

	// Number of threads per block
	blockSize = (numOfParticles_host > MaxBlockSize ? MaxBlockSize : numOfParticles_host);
	// Number of blocks (to avoid overlapping)
	numBlocks = (numOfParticles_host + blockSize - 1) / blockSize;

	double3 x_offset_host = make_double3(lambda_host / 2, 0, 0);
	double3 y_offset_host = make_double3(0, lambda_host / 2, 0);
	double3 z_offset_host = make_double3(0, 0, lambda_host / 2);
	hipMemcpyToSymbol(HIP_SYMBOL(x_offset), &x_offset_host, sizeof(double3));
	hipMemcpyToSymbol(HIP_SYMBOL(y_offset), &y_offset_host, sizeof(double3));
	hipMemcpyToSymbol(HIP_SYMBOL(z_offset), &z_offset_host, sizeof(double3));

	double3 zeroVector = make_double3(0, 0, 0);
	velocities.resize(numOfParticles_host);
	thrust::fill(thrust::device, velocities.begin(), velocities.end(), zeroVector);
	forces.resize(numOfParticles_host);
	thrust::fill(thrust::device, forces.begin(), forces.end(), zeroVector);
	taus.resize(numOfParticles_host);
	thrust::fill(thrust::device, taus.begin(), taus.end(), avgtau_host);
	hashTable.resize(hashBinsMaxSize_host * hashBinsNum_host);
	thrust::fill(thrust::device, hashTable.begin(), hashTable.end(), 0);
	hashCounts.resize(hashBinsNum_host);
	thrust::fill(thrust::device, hashCounts.begin(), hashCounts.end(), 0);
}


void iterate(
	thrust::host_vector<double3>& positionsHost,
	thrust::device_vector<double3>& positions,
	thrust::device_vector<double3>& velocities,
	thrust::device_vector<double3>& forces,
	thrust::host_vector<int>& adjsHost,
	thrust::device_vector<int>& adjs,
	thrust::device_vector<unsigned int>& adjsCounts,
	thrust::device_vector<unsigned int>& adjsStarts,
	thrust::device_vector<double>& restLengths,
	thrust::device_vector<double>& taus,
	thrust::device_vector<unsigned int>& hashTable,
	thrust::device_vector<unsigned int>& hashCounts,
	unsigned int numberOfIterations,
	double particleMass,
	double particleMassInv)
{
	// Get raw pointers to pass to kernels
	unsigned int* hashTable_ptr = thrust::raw_pointer_cast(hashTable.data());
	unsigned int* hashCounts_ptr = thrust::raw_pointer_cast(hashCounts.data());
	double3* positions_ptr = thrust::raw_pointer_cast(positions.data());
	double3* forces_ptr = thrust::raw_pointer_cast(forces.data());
	double3* velocities_ptr = thrust::raw_pointer_cast(velocities.data());
	int* adjs_ptr = thrust::raw_pointer_cast(adjs.data());
	unsigned int* adjsCounts_ptr = thrust::raw_pointer_cast(adjsCounts.data());
	unsigned int* adjsStarts_ptr = thrust::raw_pointer_cast(adjsStarts.data());
	double* restLengths_ptr = thrust::raw_pointer_cast(restLengths.data());
	double* taus_ptr = thrust::raw_pointer_cast(taus.data());

	for (int i = 0; i < numberOfIterations; ++i) {
		// Initialize hash bins for the next iteration
		initializeKernel<<<numBlocks, blockSize>>>(hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Collect - assign each particle to hash bin
		collectKernel<<<numBlocks, blockSize>>>(positions_ptr, hashTable_ptr, hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Detect collisions and compute response
		detectCollisionsKernel<<<numBlocks, blockSize>>>(positions_ptr, forces_ptr, hashTable_ptr, hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Advance velocities by half-time step (first Velocity-Verlet update)
		advanceVelocitiesKernel<<<numBlocks, blockSize>>>(velocities_ptr, forces_ptr, particleMassInv);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Advance positions by one timestep (second Velocity-Verlet update
		advancePositionsKernel<<<numBlocks, blockSize>>>(positions_ptr, velocities_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Add body forces: gravity + collision with the floor
		addBodyForcesKernel<<<numBlocks, blockSize>>>(positions_ptr, velocities_ptr, forces_ptr, particleMass);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Add spring forces
		addSpringForcesKernel<<<numBlocks, blockSize>>>(
			positions_ptr, forces_ptr, adjs_ptr, adjsCounts_ptr, adjsStarts_ptr, restLengths_ptr, taus_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// Advance velocities (third Velocitiy-Verlet update)
		advanceVelocitiesKernel<<<numBlocks, blockSize>>>(velocities_ptr, forces_ptr, particleMassInv);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

	}

	// Synchronize GPU and CPU before copying the data back
	gpuErrchk(hipDeviceSynchronize());

	thrust::copy(positions.begin(), positions.end(), positionsHost.begin());
	thrust::copy(adjs.begin(), adjs.end(), adjsHost.begin());
}


void resetVelocitiesAndForces(
	thrust::device_vector<double3>& velocities,
	thrust::device_vector<double3>& forces
)
{
	double3 zeroVector = make_double3(0, 0, 0);
	thrust::fill(thrust::device, velocities.begin(), velocities.end(), zeroVector);
	thrust::fill(thrust::device, forces.begin(), forces.end(), zeroVector);
}

/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// End of simulation interface //////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Start of kernel unit-testing helpers /////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////


	void* allocateDeviceMemory(unsigned int size)
	{
		void* ptr;
		hipMalloc(&ptr, size);
		return ptr;
	}

	void freeDeviceMemory(void* ptr) {
		hipFree(ptr);
	}

	void copyToDevice(void* devPtr, void* dataPtr, unsigned int size) {
		hipMemcpy(devPtr, dataPtr, size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
	}

	void copyFromDevice(void* dataPtr, void* devPtr, unsigned int size) {
		hipMemcpy(dataPtr, devPtr, size, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
	}

	void initialize(
		thrust::device_vector<unsigned int>& hashCounts) 
	{
		unsigned int* hashCounts_ptr = thrust::raw_pointer_cast(&hashCounts[0]);
		initializeKernel<<<numBlocks, blockSize>>>(hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void collect(
		thrust::device_vector<double3>& positions,
		thrust::device_vector<unsigned int>& hashTable,
		thrust::device_vector<unsigned int>& hashCounts)
	{
		double3* positions_ptr = thrust::raw_pointer_cast(&positions[0]);
		unsigned int* hashTable_ptr = thrust::raw_pointer_cast(&hashTable[0]);
		unsigned int* hashCounts_ptr = thrust::raw_pointer_cast(&hashCounts[0]);
		collectKernel<<<numBlocks, blockSize>>>(positions_ptr, hashTable_ptr, hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void detectCollisions(
		thrust::device_vector<double3>& positions,
		thrust::device_vector<double3>& forces,
		thrust::device_vector<unsigned int>& hashTable,
		thrust::device_vector<unsigned int>& hashCounts)
	{
		double3* positions_ptr = thrust::raw_pointer_cast(positions.data());
		double3* forces_ptr = thrust::raw_pointer_cast(forces.data());
		unsigned int* hashTable_ptr = thrust::raw_pointer_cast(hashTable.data());
		unsigned int* hashCounts_ptr = thrust::raw_pointer_cast(hashCounts.data());
		detectCollisionsKernel<<<numBlocks, blockSize>>>(positions_ptr, forces_ptr, hashTable_ptr, hashCounts_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void advanceVelocities(
		thrust::device_vector<double3>& velocities,
		thrust::device_vector<double3>& forces,
		double massInv)
	{
		double3* velocities_ptr = thrust::raw_pointer_cast(&velocities[0]);
		double3* forces_ptr = thrust::raw_pointer_cast(&forces[0]);
		advanceVelocitiesKernel<<<numBlocks, blockSize>>>(velocities_ptr, forces_ptr, massInv);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void advancePositions(
		thrust::device_vector<double3>& positions,
		thrust::device_vector<double3>& velocities)
	{
		double3* positions_ptr = thrust::raw_pointer_cast(&positions[0]);
		double3* velocities_ptr = thrust::raw_pointer_cast(&velocities[0]);
		advancePositionsKernel<<<numBlocks, blockSize>>>(positions_ptr, velocities_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void addBodyForces(
		thrust::device_vector<double3>& positions,
		thrust::device_vector<double3>& velocities,
		thrust::device_vector<double3>& forces,
		double mass)
	{
		double3* positions_ptr = thrust::raw_pointer_cast(&positions[0]);
		double3* velocities_ptr = thrust::raw_pointer_cast(&velocities[0]);
		double3* forces_ptr = thrust::raw_pointer_cast(&forces[0]);
		addBodyForcesKernel<<<numBlocks, blockSize>>>(positions_ptr, velocities_ptr, forces_ptr, mass);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	void addSpringForces(
		thrust::device_vector<double3>& positions,
		thrust::device_vector<double3>& forces,
		thrust::device_vector<int>& adjs,
		thrust::device_vector<unsigned int>& adjsCounts,
		thrust::device_vector<unsigned int>& adjsStarts,
		thrust::device_vector<double>& restLengths,
		thrust::device_vector<double>& taus) 
	{
		double3* positions_ptr = thrust::raw_pointer_cast(&positions[0]);
		double3* forces_ptr = thrust::raw_pointer_cast(&forces[0]);
		int* adjs_ptr = thrust::raw_pointer_cast(&adjs[0]);
		unsigned int* adjsCounts_ptr = thrust::raw_pointer_cast(&adjsCounts[0]);
		unsigned int* adjsStarts_ptr = thrust::raw_pointer_cast(&adjsStarts[0]);
		double* restLengths_ptr = thrust::raw_pointer_cast(&restLengths[0]);
		double* taus_ptr = thrust::raw_pointer_cast(&taus[0]);
		addSpringForcesKernel<<<numBlocks, blockSize>>>(
			positions_ptr, forces_ptr, adjs_ptr, adjsCounts_ptr, adjsStarts_ptr, restLengths_ptr, taus_ptr);
#if ERRCHECK_AND_SYNC
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

/////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////// End of kernel unit-testing helpers ///////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////


}