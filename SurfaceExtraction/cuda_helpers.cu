#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ __forceinline__ float3 operator+(const float3& a, const float3& b)
{
   float3 out;
   out.x = a.x + b.x;
   out.y = a.y + b.y;
   out.z = a.z + b.z;
   return out;
}
__device__ __forceinline__ float3 operator-(const float3& a, const float3& b)
{
   float3 out;
   out.x = a.x - b.x;
   out.y = a.y - b.y;
   out.z = a.z - b.z;
   return out;
}
__device__ __forceinline__ float3 operator*(const double b, const float3& a)
{
   float3 out;
   out.x = a.x * b;
   out.y = a.y * b;
   out.z = a.z * b;
   return out;
}
__device__ __forceinline__ float3& operator+=(float3& lhs, const float3& rhs)
{
   lhs.x += rhs.x;
   lhs.y += rhs.y;
   lhs.z += rhs.z;
   return lhs;
}
__device__ __forceinline__ float3& operator-=(float3& lhs, const float3& rhs)
{
   lhs.x -= rhs.x;
   lhs.y -= rhs.y;
   lhs.z -= rhs.z;
   return lhs;
}
__device__ __forceinline__ float3& operator*=(float3& lhs, const double rhs)
{
   lhs.x *= rhs;
   lhs.y *= rhs;
   lhs.z *= rhs;
   return lhs;
}
__device__ __forceinline__ float dot(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}
__device__ __forceinline__ float3 normalize(float3 v)
{
    float invLen = rsqrtf(dot(v, v));
    return invLen * v;
}
__device__ __forceinline__ float3 cross(float3 a, float3 b)
{  float3 out;

   out.x = a.y * b.z - b.y * a.z;
   out.y = a.z * b.x - b.z * a.x;
   out.z = a.x * b.y - b.x * a.y;
   return out;
}

#define EPSILON  1e-5
#define DOUBLE_EPSILON  1e-15

// surface_midpoints and weights are used for computing all points that
// may occur on the marched surface inside a tet. (total points = 44)
__constant__ unsigned char surface_midpoints[][5] = {
   /* {cnt, vertex list} */
   {1, 0},     {1, 1},     {1, 2},     {1, 3},
   {2, 0,1},   {2, 0,2},   {2, 0,3},   {2, 1,0},   {2, 1,2},   {2, 1,3},
   {2, 2,0},   {2, 2,1},   {2, 2,3},   {2, 3,0},   {2, 3,1},   {2, 3,2},
   {3, 0,1,2}, {3, 0,1,3}, {3, 0,2,1}, {3, 0,2,3}, {3, 0,3,1},
   {3, 0,3,2}, {3, 1,0,2}, {3, 1,0,3}, {3, 1,2,0}, {3, 1,2,3},
   {3, 1,3,0}, {3, 1,3,2}, {3, 2,0,1}, {3, 2,0,3}, {3, 2,1,0},
   {3, 2,1,3}, {3, 2,3,0}, {3, 2,3,1}, {3, 3,0,1}, {3, 3,0,2},
   {3, 3,1,0}, {3, 3,1,2}, {3, 3,2,0}, {3, 3,2,1},
   {4, 0,1,2,3},{4, 1,0,2,3},{4, 2,0,1,3},{4, 3,0,1,2},

   // another set of different weight vertices
   {2, 0,1},   {2, 0,2},   {2, 0,3},   {2, 1,0},   {2, 1,2},   {2, 1,3},
   {2, 2,0},   {2, 2,1},   {2, 2,3},   {2, 3,0},   {2, 3,1},   {2, 3,2},
   {3, 0,1,2}, {3, 0,1,3}, {3, 0,2,1}, {3, 0,2,3}, {3, 0,3,1},
   {3, 0,3,2}, {3, 1,0,2}, {3, 1,0,3}, {3, 1,2,0}, {3, 1,2,3},
   {3, 1,3,0}, {3, 1,3,2}, {3, 2,0,1}, {3, 2,0,3}, {3, 2,1,0},
   {3, 2,1,3}, {3, 2,3,0}, {3, 2,3,1}, {3, 3,0,1}, {3, 3,0,2},
   {3, 3,1,0}, {3, 3,1,2}, {3, 3,2,0}, {3, 3,2,1},
   {4, 0,1,2,3},{4, 1,0,2,3},{4, 2,0,1,3},{4, 3,0,1,2},

   // for switching dominant weights
   {3, 0,1,2}, {3, 0,1,3}, {3, 0,2,1}, {3, 0,2,3}, {3, 0,3,1},
   {3, 0,3,2}, {3, 1,0,2}, {3, 1,0,3}, {3, 1,2,0}, {3, 1,2,3},
   {3, 1,3,0}, {3, 1,3,2}, {3, 2,0,1}, {3, 2,0,3}, {3, 2,1,0},
   {3, 2,1,3}, {3, 2,3,0}, {3, 2,3,1}, {3, 3,0,1}, {3, 3,0,2},
   {3, 3,1,0}, {3, 3,1,2}, {3, 3,2,0}, {3, 3,2,1},
};
#define  N_MIDPOINTS  (sizeof(surface_midpoints)/sizeof(surface_midpoints[0]))
#define  w2_2_a    4.1/9.
#define  w2_2_b    2.45/9.
#define  w2_2_c    (1.-w2_2_a-w2_2_b)
#define  w2_3_a    3./9.
#define  w2_3_b    3./9.
#define  w2_3_c    (1.-w2_3_a-w2_3_b)

// __constant__ float weight1_1[] = {1.};
// __constant__ float weight1_2[] = {4.7/9.,4.3/9.};
// __constant__ float weight1_3[] = {3.2/9.,2.9/9.,2.9/9.};
// __constant__ float weight1_4[] = {0.28,0.24,0.24,0.24};
// __constant__ float weight2_1[] = {1.};
// __constant__ float weight2_2[] = {4.7/9.,4.3/9.};
// __constant__ float weight2_3[] = {w2_2_a, w2_2_b, w2_2_c};
// __constant__ float weight2_4[] = {0.25,0.25,0.25,0.25};
// __constant__ float weight3_3[] = {w2_2_b, w2_2_a, w2_2_c};

__constant__ float weight1_1[] = {1.};
__constant__ float weight1_2[] = {4.5/9.,4.5/9.};
__constant__ float weight1_3[] = {3./9.,3./9.,3./9.};
__constant__ float weight1_4[] = {0.25,0.25,0.25,0.25};
// another set of weights
__constant__ float weight2_1[] = {1.};
__constant__ float weight2_2[] = {4.5/9.,4.5/9.};
   // weight2_3 used in one broken edge cases
__constant__ float weight2_3[] = {1.4/9.,3.8/9.,3.8/9.};
__constant__ float weight2_4[] = {0.25,0.25,0.25,0.25};
// used for switching the dominant vertex
__constant__ float weight3_3[] = {w2_3_a, w2_3_b, w2_3_c};

__constant__ float *surface_weights[] = {
   weight1_1, weight1_1, weight1_1, weight1_1,
   weight1_2, weight1_2, weight1_2, weight1_2, weight1_2, weight1_2,
   weight1_2, weight1_2, weight1_2, weight1_2, weight1_2, weight1_2,
   weight1_3, weight1_3, weight1_3, weight1_3, weight1_3, weight1_3,
   weight1_3, weight1_3, weight1_3, weight1_3, weight1_3, weight1_3,
   weight1_3, weight1_3, weight1_3, weight1_3, weight1_3, weight1_3,
   weight1_3, weight1_3, weight1_3, weight1_3, weight1_3, weight1_3,
   weight1_4, weight1_4, weight1_4, weight1_4,

   weight2_2, weight2_2, weight2_2, weight2_2, weight2_2, weight2_2,
   weight2_2, weight2_2, weight2_2, weight2_2, weight2_2, weight2_2,
   weight2_3, weight2_3, weight2_3, weight2_3, weight2_3, weight2_3,
   weight2_3, weight2_3, weight2_3, weight2_3, weight2_3, weight2_3,
   weight2_3, weight2_3, weight2_3, weight2_3, weight2_3, weight2_3,
   weight2_3, weight2_3, weight2_3, weight2_3, weight2_3, weight2_3,
   weight2_4, weight2_4, weight2_4, weight2_4,

   weight3_3, weight3_3, weight3_3, weight3_3, weight3_3, weight3_3,
   weight3_3, weight3_3, weight3_3, weight3_3, weight3_3, weight3_3,
   weight3_3, weight3_3, weight3_3, weight3_3, weight3_3, weight3_3,
   weight3_3, weight3_3, weight3_3, weight3_3, weight3_3, weight3_3,
};

// surface_** give a list of inner triangles.
// boundary_surface_**_# contain lists of boundary triangles.
// v_compute_** give us a list of midpoints needed to be computed for
//    composing surface triangles.
// where ** indicates a case number, and # indicates a face number.
// An order of face numbers follows data in face_check.
// A face_check is used for checking a boundary face.
__constant__ unsigned char face_check[4][3] = {
   {0,2,1}, {0,1,3}, {1,2,3}, {0,3,2}};
#include "surface_array.h"

__constant__ unsigned char *inner_surface[64] = {
   surface_00, surface_01, surface_02, surface_03, surface_04, surface_05,
   surface_06, surface_07, surface_08, surface_09, surface_10, surface_11,
   surface_12, surface_13, surface_14, surface_15, surface_16, surface_17,
   surface_18, surface_19, surface_20, surface_21, surface_22, surface_23,
   surface_24, surface_25, surface_26, surface_27, surface_28, surface_29,
   surface_30, surface_31, surface_32, surface_33, surface_34, surface_35,
   surface_36, surface_37, surface_38, surface_39, surface_40, surface_41,
   surface_42, surface_43, surface_44, surface_45, surface_46, surface_47,
   surface_48, surface_49, surface_50, surface_51, surface_52, surface_53,
   surface_54, surface_55, surface_56, surface_57, surface_58, surface_59,
   surface_60, surface_61, surface_62, surface_63, };
__constant__ unsigned char *boundary_surface[64][4] = {
   boundary_surface_00_0, boundary_surface_00_1, boundary_surface_00_2,
   boundary_surface_00_3, boundary_surface_01_0, boundary_surface_01_1,
   boundary_surface_01_2, boundary_surface_01_3, boundary_surface_02_0,
   boundary_surface_02_1, boundary_surface_02_2, boundary_surface_02_3,
   boundary_surface_03_0, boundary_surface_03_1, boundary_surface_03_2,
   boundary_surface_03_3, boundary_surface_04_0, boundary_surface_04_1,
   boundary_surface_04_2, boundary_surface_04_3, boundary_surface_05_0,
   boundary_surface_05_1, boundary_surface_05_2, boundary_surface_05_3,
   boundary_surface_06_0, boundary_surface_06_1, boundary_surface_06_2,
   boundary_surface_06_3, boundary_surface_07_0, boundary_surface_07_1,
   boundary_surface_07_2, boundary_surface_07_3, boundary_surface_08_0,
   boundary_surface_08_1, boundary_surface_08_2, boundary_surface_08_3,
   boundary_surface_09_0, boundary_surface_09_1, boundary_surface_09_2,
   boundary_surface_09_3, boundary_surface_10_0, boundary_surface_10_1,
   boundary_surface_10_2, boundary_surface_10_3, boundary_surface_11_0,
   boundary_surface_11_1, boundary_surface_11_2, boundary_surface_11_3,
   boundary_surface_12_0, boundary_surface_12_1, boundary_surface_12_2,
   boundary_surface_12_3, boundary_surface_13_0, boundary_surface_13_1,
   boundary_surface_13_2, boundary_surface_13_3, boundary_surface_14_0,
   boundary_surface_14_1, boundary_surface_14_2, boundary_surface_14_3,
   boundary_surface_15_0, boundary_surface_15_1, boundary_surface_15_2,
   boundary_surface_15_3, boundary_surface_16_0, boundary_surface_16_1,
   boundary_surface_16_2, boundary_surface_16_3, boundary_surface_17_0,
   boundary_surface_17_1, boundary_surface_17_2, boundary_surface_17_3,
   boundary_surface_18_0, boundary_surface_18_1, boundary_surface_18_2,
   boundary_surface_18_3, boundary_surface_19_0, boundary_surface_19_1,
   boundary_surface_19_2, boundary_surface_19_3, boundary_surface_20_0,
   boundary_surface_20_1, boundary_surface_20_2, boundary_surface_20_3,
   boundary_surface_21_0, boundary_surface_21_1, boundary_surface_21_2,
   boundary_surface_21_3, boundary_surface_22_0, boundary_surface_22_1,
   boundary_surface_22_2, boundary_surface_22_3, boundary_surface_23_0,
   boundary_surface_23_1, boundary_surface_23_2, boundary_surface_23_3,
   boundary_surface_24_0, boundary_surface_24_1, boundary_surface_24_2,
   boundary_surface_24_3, boundary_surface_25_0, boundary_surface_25_1,
   boundary_surface_25_2, boundary_surface_25_3, boundary_surface_26_0,
   boundary_surface_26_1, boundary_surface_26_2, boundary_surface_26_3,
   boundary_surface_27_0, boundary_surface_27_1, boundary_surface_27_2,
   boundary_surface_27_3, boundary_surface_28_0, boundary_surface_28_1,
   boundary_surface_28_2, boundary_surface_28_3, boundary_surface_29_0,
   boundary_surface_29_1, boundary_surface_29_2, boundary_surface_29_3,
   boundary_surface_30_0, boundary_surface_30_1, boundary_surface_30_2,
   boundary_surface_30_3, boundary_surface_31_0, boundary_surface_31_1,
   boundary_surface_31_2, boundary_surface_31_3, boundary_surface_32_0,
   boundary_surface_32_1, boundary_surface_32_2, boundary_surface_32_3,
   boundary_surface_33_0, boundary_surface_33_1, boundary_surface_33_2,
   boundary_surface_33_3, boundary_surface_34_0, boundary_surface_34_1,
   boundary_surface_34_2, boundary_surface_34_3, boundary_surface_35_0,
   boundary_surface_35_1, boundary_surface_35_2, boundary_surface_35_3,
   boundary_surface_36_0, boundary_surface_36_1, boundary_surface_36_2,
   boundary_surface_36_3, boundary_surface_37_0, boundary_surface_37_1,
   boundary_surface_37_2, boundary_surface_37_3, boundary_surface_38_0,
   boundary_surface_38_1, boundary_surface_38_2, boundary_surface_38_3,
   boundary_surface_39_0, boundary_surface_39_1, boundary_surface_39_2,
   boundary_surface_39_3, boundary_surface_40_0, boundary_surface_40_1,
   boundary_surface_40_2, boundary_surface_40_3, boundary_surface_41_0,
   boundary_surface_41_1, boundary_surface_41_2, boundary_surface_41_3,
   boundary_surface_42_0, boundary_surface_42_1, boundary_surface_42_2,
   boundary_surface_42_3, boundary_surface_43_0, boundary_surface_43_1,
   boundary_surface_43_2, boundary_surface_43_3, boundary_surface_44_0,
   boundary_surface_44_1, boundary_surface_44_2, boundary_surface_44_3,
   boundary_surface_45_0, boundary_surface_45_1, boundary_surface_45_2,
   boundary_surface_45_3, boundary_surface_46_0, boundary_surface_46_1,
   boundary_surface_46_2, boundary_surface_46_3, boundary_surface_47_0,
   boundary_surface_47_1, boundary_surface_47_2, boundary_surface_47_3,
   boundary_surface_48_0, boundary_surface_48_1, boundary_surface_48_2,
   boundary_surface_48_3, boundary_surface_49_0, boundary_surface_49_1,
   boundary_surface_49_2, boundary_surface_49_3, boundary_surface_50_0,
   boundary_surface_50_1, boundary_surface_50_2, boundary_surface_50_3,
   boundary_surface_51_0, boundary_surface_51_1, boundary_surface_51_2,
   boundary_surface_51_3, boundary_surface_52_0, boundary_surface_52_1,
   boundary_surface_52_2, boundary_surface_52_3, boundary_surface_53_0,
   boundary_surface_53_1, boundary_surface_53_2, boundary_surface_53_3,
   boundary_surface_54_0, boundary_surface_54_1, boundary_surface_54_2,
   boundary_surface_54_3, boundary_surface_55_0, boundary_surface_55_1,
   boundary_surface_55_2, boundary_surface_55_3, boundary_surface_56_0,
   boundary_surface_56_1, boundary_surface_56_2, boundary_surface_56_3,
   boundary_surface_57_0, boundary_surface_57_1, boundary_surface_57_2,
   boundary_surface_57_3, boundary_surface_58_0, boundary_surface_58_1,
   boundary_surface_58_2, boundary_surface_58_3, boundary_surface_59_0,
   boundary_surface_59_1, boundary_surface_59_2, boundary_surface_59_3,
   boundary_surface_60_0, boundary_surface_60_1, boundary_surface_60_2,
   boundary_surface_60_3, boundary_surface_61_0, boundary_surface_61_1,
   boundary_surface_61_2, boundary_surface_61_3, boundary_surface_62_0,
   boundary_surface_62_1, boundary_surface_62_2, boundary_surface_62_3,
   boundary_surface_63_0, boundary_surface_63_1, boundary_surface_63_2,
   boundary_surface_63_3, };
__constant__ unsigned char *v_compute[64] = {
   v_compute_00, v_compute_01, v_compute_02, v_compute_03, v_compute_04,
   v_compute_05, v_compute_06, v_compute_07, v_compute_08, v_compute_09,
   v_compute_10, v_compute_11, v_compute_12, v_compute_13, v_compute_14,
   v_compute_15, v_compute_16, v_compute_17, v_compute_18, v_compute_19,
   v_compute_20, v_compute_21, v_compute_22, v_compute_23, v_compute_24,
   v_compute_25, v_compute_26, v_compute_27, v_compute_28, v_compute_29,
   v_compute_30, v_compute_31, v_compute_32, v_compute_33, v_compute_34,
   v_compute_35, v_compute_36, v_compute_37, v_compute_38, v_compute_39,
   v_compute_40, v_compute_41, v_compute_42, v_compute_43, v_compute_44,
   v_compute_45, v_compute_46, v_compute_47, v_compute_48, v_compute_49,
   v_compute_50, v_compute_51, v_compute_52, v_compute_53, v_compute_54,
   v_compute_55, v_compute_56, v_compute_57, v_compute_58, v_compute_59,
   v_compute_60, v_compute_61, v_compute_62, v_compute_63, };

__constant__ unsigned char edge_connected_points[12] = {
   0,1,  0,2,  0,3,  1,2,  1,3,  2,3 };

// edge_cut_points contain a list of (x,y) where x indicates the "edge" element
// number in surface_midpoints that has a y end-point vertices.
// y refers to an element in edge_connected_points; y >= 6 indicates the
// reverse order of the end points.
__constant__ unsigned char edge_cut_points[][2] = {
   { 4, 0},   { 5, 1},   { 6, 2},   { 7, 6},   { 8, 3},   { 9, 4},
   {10, 7},   {11, 9},   {12, 5},   {13, 8},   {14,10},   {15,11},
   {44, 0},   {45, 1},   {46, 2},   {47, 6},   {48, 3},   {49, 4},
   {50, 7},   {51, 9},   {52, 5},   {53, 8},   {54,10},   {55,11}};

__device__ __forceinline__ void vertex_xform(float3 *vout,
   float3 vin, float3 T0, float3 T1, float *R)
{  float3 t;

   t = vin;
   t -= T0;
   (*vout).x = R[0]*t.x + R[3]*t.y + R[6]*t.z;
   (*vout).y = R[1]*t.x + R[4]*t.y + R[7]*t.z;
   (*vout).z = R[2]*t.x + R[5]*t.y + R[8]*t.z;
   (*vout) += T1;
}

__device__ __forceinline__ int is_broken(int b1, int b2, float3 p1, float3 p2,
   float3 q1, float3 q2, float broken_distance)
{  float3 diff;
   float d1, d2;

   if (b1 != b2)
      return 1;
   diff = p1 - p2;
   d1 = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
   diff = q1 - q2;
   d2 = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
   // if ((d2-d1)/d1 < broken_distance)
   //    return 0;
   // else
   //    return 1;
   if (d2-d1 < broken_distance)
      return 0;
   else
      return 1;
}

__device__ __forceinline__ float breakage(float3 p1, float3 p2,
   float3 q1, float3 q2)
{  float3 diff;
   float d1, d2;

   diff = p1 - p2;
   d1 = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
   diff = q1 - q2;
   d2 = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
   //return (d2-d1)/d1;
   return (d2-d1);
}

__device__ __forceinline__ double mat_det(float *m)
{
   return  +(double)m[0]*(m[4]*m[8]-m[7]*m[5])
           -(double)m[1]*(m[3]*m[8]-m[6]*m[5])
           +(double)m[2]*(m[3]*m[7]-m[6]*m[4]);
}

__device__ __forceinline__ void compute_normals(float3 *nout, float3 *vin)
{  float3 e0, e1, n;

   e0 = vin[0] - vin[1];
   e1 = vin[2] - vin[1];
   e0 = normalize(e0);
   e1 = normalize(e1);
   n  = cross(e0, e1);
   nout[0] = n;
   nout[1] = n;
   nout[2] = n;
}

#define DAMAGE  0.01
__device__ __forceinline__ float cut_function(float3 p, float *cut_plane)
{
   return cut_plane[0]*p.x + cut_plane[1]*p.y + cut_plane[2]*p.z +
      cut_plane[3];
   //return 0.25*sin(p.x*2*M_PI)+0.5-p.y;
   //return n.x*p.x + n.y*p.y + n.z*p.z - 0.35;
}

// Compute cut weights between cut_function and tetrahedral edges.
__global__ void compute_cut_weights(float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ tetras, int n_tetras,
   float* __restrict__ cut_weights, unsigned char* __restrict__ broken_edges,
   float* cut_plane)
{  int block_id, idx, j, k, l, part_id;
   float3 t_vertices[4], t_sim_vertices[4], *p, *s;
   unsigned char *e, *ecp;
   float *cw, d1, d2;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
   }

   cw = cut_weights+6*idx;
   e = broken_edges+6*idx;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;
   for (j=0; j < 6; j++) {
      k = ecp[2*j];
      l = ecp[2*j+1];
      d1 = cut_function(p[k], cut_plane);
      d2 = cut_function(p[l], cut_plane);
      if (d1 >=0 && d2 < 0 || d1 < 0 && d2 >= 0) {
         cw[j] = -d2/(d1-d2);
         e[j] = 1;
      }
   }
}

// Routine to extract fracture surface with sliding cut-points
__global__ void march_tetra_with_cut_weights(float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   int* __restrict__ n_bonds,
   float* __restrict__ cut_weights,
   unsigned char* __restrict__ broken_edges,
   int* __restrict__ accum_tri_counts,
   int label_component,
   float3* __restrict__ out_vertices, float3* __restrict__ out_normals,
   float3* __restrict__ out_uvs,
   int* __restrict__ out_tri_labels)
{  int block_id, idx, j, k, l, m, n;
   int label_id, part_id, point_id;
   int t_labels[4], t_part_ids[4], case_id, tri_counts;
   float3 t_vertices[4], s_vertices[N_MIDPOINTS];
   unsigned char *e, *ecp, *s_idx, **b_idx, *vc_idx, *b;
   float I[9]={1,0,0, 0,1,0, 0,0,1}, *used_R, *cw, w;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts = accum_tri_counts[idx];
   b = boundary_flags + 4*idx;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_labels[j] = labels[part_id];
      t_part_ids[j] = part_id;
   }

   e  = broken_edges+6*idx;
   cw = cut_weights+6*idx;
   case_id = 0;
   for (j=0; j < 6; j++)
      case_id |= e[j] << (5-j);

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }

   // recompute cut-points.
   ecp = edge_connected_points;
   for (j=0; j < sizeof(edge_cut_points)/sizeof(edge_cut_points[0]); j++) {
      m = edge_cut_points[j][0];
      n = edge_cut_points[j][1];
      if (n >= 6) {
         n -= 6;
         k = ecp[2*n+1];
         l = ecp[2*n];
         w = 1.-cw[n];
      }
      else {
         k = ecp[2*n];
         l = ecp[2*n+1];
         w = cw[n];
      }
      s_vertices[m] = w*t_vertices[k] + (1.-w)*t_vertices[l];
   }

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = 3*tri_counts;
      // Loop through all triangle's vertices.
      for (l=0; l < 3; l++) {
         // For s_idx[1+...], we need skip one slot (s_idx[0]),
         //    which indicates a number of triangles.
         // surf_midpoints[s_idx[1+3*k+l]][1] gives a dominant vertex.
         point_id = surface_midpoints[s_idx[1+3*k+l]][1];
         part_id = t_part_ids[point_id];
         used_R = R+9*part_id;
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         out_uvs[m+l] = s_vertices[s_idx[1+3*k+l]];
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[part_id], T1[part_id], used_R);
      }
      compute_normals(out_normals+m, out_vertices+m);
      tri_counts++;
   }

   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      if (b[j]) {
         // Loop through all boundary triangles.
         // b_idx[j] ~ a list of triangles of the face j.
         // b_idx[j][0] ~ a number of triangles of the face j.
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = 3*tri_counts;
            // Loop through all triangle's vertices.
            for (l=0; l < 3; l++) {
               point_id = surface_midpoints[b_idx[j][1+3*k+l]][1];
               part_id = t_part_ids[point_id];
               used_R = R+9*part_id;
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               out_uvs[m+l] = s_vertices[b_idx[j][1+3*k+l]];
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[part_id], T1[part_id], used_R);
            }
            compute_normals(out_normals+m, out_vertices+m);
            tri_counts++;
         }
      }
   }
}

// Routine to extract fracture surface with sliding cut-points
// Also generating triangle split side of a cutting plane
__global__ void march_tetra_with_cut_weights_and_split(
   float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   int* __restrict__ n_bonds,
   float* __restrict__ cut_weights,
   unsigned char* __restrict__ broken_edges,
   int* __restrict__ accum_tri_counts,
   int label_component,
   float3* __restrict__ out_vertices, float3* __restrict__ out_normals,
   float3* __restrict__ out_uvs,
   int* __restrict__ out_tri_labels,
   float* cut_plane, unsigned char* __restrict__ tri_cut_sides)
{  int block_id, idx, j, k, l, m, n, side0_cnt;
   int label_id, part_id, point_id;
   int t_labels[4], t_part_ids[4], case_id, tri_counts;
   float3 t_vertices[4], s_vertices[N_MIDPOINTS];
   unsigned char *e, *ecp, *s_idx, **b_idx, *vc_idx, *b;
   float I[9]={1,0,0, 0,1,0, 0,0,1}, *used_R, *cw, w, dist;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts = accum_tri_counts[idx];
   b = boundary_flags + 4*idx;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_labels[j] = labels[part_id];
      t_part_ids[j] = part_id;
   }

   e  = broken_edges+6*idx;
   cw = cut_weights+6*idx;
   case_id = 0;
   for (j=0; j < 6; j++)
      case_id |= e[j] << (5-j);

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }

   // recompute cut-points.
   ecp = edge_connected_points;
   for (j=0; j < sizeof(edge_cut_points)/sizeof(edge_cut_points[0]); j++) {
      m = edge_cut_points[j][0];
      n = edge_cut_points[j][1];
      if (n >= 6) {
         n -= 6;
         k = ecp[2*n+1];
         l = ecp[2*n];
         w = 1.-cw[n];
      }
      else {
         k = ecp[2*n];
         l = ecp[2*n+1];
         w = cw[n];
      }
      s_vertices[m] = (w+0.0001)*t_vertices[k] + (1.-w-0.0001)*t_vertices[l];
   }

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = 3*tri_counts;
      // Loop through all triangle's vertices.
      side0_cnt = 0;
      for (l=0; l < 3; l++) {
         // For s_idx[1+...], we need skip one slot (s_idx[0]),
         //    which indicates a number of triangles.
         // surf_midpoints[s_idx[1+3*k+l]][1] gives a dominant vertex.
         point_id = surface_midpoints[s_idx[1+3*k+l]][1];
         part_id = t_part_ids[point_id];
         used_R = R+9*part_id;
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         out_uvs[m+l] = s_vertices[s_idx[1+3*k+l]];
         dist = cut_function(s_vertices[s_idx[1+3*k+l]], cut_plane);
         if (dist < 0)
            side0_cnt++;
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[part_id], T1[part_id], used_R);
      }
      if (side0_cnt > 0)
         tri_cut_sides[m] = tri_cut_sides[m+1] = tri_cut_sides[m+2] = 0;
      else
         tri_cut_sides[m] = tri_cut_sides[m+1] = tri_cut_sides[m+2] = 1;
      compute_normals(out_normals+m, out_vertices+m);
      tri_counts++;
   }

   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      if (b[j]) {
         // Loop through all boundary triangles.
         // b_idx[j] ~ a list of triangles of the face j.
         // b_idx[j][0] ~ a number of triangles of the face j.
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = 3*tri_counts;
            // Loop through all triangle's vertices.
            side0_cnt = 0;
            for (l=0; l < 3; l++) {
               point_id = surface_midpoints[b_idx[j][1+3*k+l]][1];
               part_id = t_part_ids[point_id];
               used_R = R+9*part_id;
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               out_uvs[m+l] = s_vertices[b_idx[j][1+3*k+l]];
               dist = cut_function(s_vertices[b_idx[j][1+3*k+l]], cut_plane);
               if (dist < 0)
                  side0_cnt++;
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[part_id], T1[part_id], used_R);
            }
            if (side0_cnt > 0)
               tri_cut_sides[m] = tri_cut_sides[m+1] = tri_cut_sides[m+2] = 0;
            else
               tri_cut_sides[m] = tri_cut_sides[m+1] = tri_cut_sides[m+2] = 1;
            compute_normals(out_normals+m, out_vertices+m);
            tri_counts++;
         }
      }
   }
}

// similar to march_tetra() except combining all routines into one function.
__global__ void march_tetra_cuts(float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   // tetras contains a list of particle ids of tets.
   // bounders contains a list of opposite tet ids of a tet's vertex?
   int* __restrict__ tetras, int* __restrict__ bounders,
   int n_tetras, int* __restrict__ n_bonds,
   float* __restrict__ cut_weights,
   unsigned char* __restrict__ broken_edges, float broken_distance,
   int label_component,
   float3* __restrict__ out_vertices, int* __restrict__ tri_counts,
   int* __restrict__ out_tri_labels,
   int tris_per_tet)
{  int block_id, idx, j, k, l, m, n, bnd_found;
   int label_id, part_id, point_id;
   int t_labels[4], t_part_ids[4], case_id, *t, bnd0, bnd1, bnd2;
   int chk_list[12], chk_count;
   float3 t_vertices[4], t_sim_vertices[4], s_vertices[N_MIDPOINTS], *p, *s;
   unsigned char *e, *ecp, *s_idx, **b_idx, *vc_idx;
   float I[9]={1,0,0, 0,1,0, 0,0,1}, *used_R, *cw, w;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts[idx] = 0;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
      t_labels[j] = labels[part_id];
      t_part_ids[j] = part_id;
   }

   cw = cut_weights+6*idx;
   e = broken_edges+6*idx;
   t = t_labels;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;

   case_id = 0;
   for (j=0; j < 6; j++) {
      k = ecp[2*j];
      l = ecp[2*j+1];
      e[j] |= is_broken(t[k], t[l], p[k], p[l], s[k], s[l], broken_distance);
      case_id |= e[j] << (5-j);
   }

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }

   // recompute cut-points.
   for (j=0; j < sizeof(edge_cut_points)/sizeof(edge_cut_points[0]); j++) {
      m = edge_cut_points[j][0];
      n = edge_cut_points[j][1];
      if (n >= 6) {
         n -= 6;
         k = ecp[2*n+1];
         l = ecp[2*n];
         w = 1.-cw[n];
      }
      else {
         k = ecp[2*n];
         l = ecp[2*n+1];
         w = cw[n];
      }
      s_vertices[m] = (w+0.0001)*t_vertices[k] + (1.-w)*t_vertices[l];
   }

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = idx*tris_per_tet*3 + 3*tri_counts[idx];
      // Loop through all triangle's vertices.
      for (l=0; l < 3; l++) {
         // For s_idx[1+...], we need skip one slot (s_idx[0]),
         //    which indicates a number of triangles.
         // surf_midpoints[s_idx[1+3*k+l]][1] gives a dominant vertex.
         point_id = surface_midpoints[s_idx[1+3*k+l]][1];
         part_id = t_part_ids[point_id];
         used_R = R+9*part_id;
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[part_id], T1[part_id], used_R);
      }
      tri_counts[idx]++;
   }

   chk_count = 0;
   // Loop through all vertices of the tet in the mesh file.
   //    If a boundary value of a vertex is -1, store the corresponding
   //    particle id (assumed to be on a boundary) in chk_list.
   // See also a mesh file format.
   for (j=0; j < 4; j++) {
      if (bounders[idx*4 + j] == -1)
         for (k=0; k < 4; k++)
            if (k != j)
               chk_list[chk_count++] = tetras[idx*4 + k];
   }
   if (chk_count == 0)
      return;
   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      // bnd0,1,2 are particle ids of a face.
      bnd0 = tetras[idx*4 + face_check[j][0]];
      bnd1 = tetras[idx*4 + face_check[j][1]];
      bnd2 = tetras[idx*4 + face_check[j][2]];
      bnd_found = 0;
      // If all 3 particle ids from a face we are about to generate are the
      //    same as ones from the mesh file, these particles are on a boundary.
      for (k=0; k < chk_count; k+=3)
         if ((bnd0==chk_list[k]||bnd0==chk_list[k+1]||bnd0==chk_list[k+2]) &&
             (bnd1==chk_list[k]||bnd1==chk_list[k+1]||bnd1==chk_list[k+2]) &&
             (bnd2==chk_list[k]||bnd2==chk_list[k+1]||bnd2==chk_list[k+2])) {
            bnd_found = 1;
            break;
         }
      // Generating boundary triangles.
      if (bnd_found == 1) {
         // Loop through all boundary triangles.
         // b_idx[j] ~ a list of triangles of the face j.
         // b_idx[j][0] ~ a number of triangles of the face j.
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = idx*tris_per_tet*3 + 3*tri_counts[idx];
            // Loop through all triangle's vertices.
            for (l=0; l < 3; l++) {
               point_id = surface_midpoints[b_idx[j][1+3*k+l]][1];
               part_id = t_part_ids[point_id];
               used_R = R+9*part_id;
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[part_id], T1[part_id], used_R);
            }
            tri_counts[idx]++;
         }
      }
   }
}

// Standard routine to extract fracture surface
// using per particle transformation
__global__ void march_tetra(float3* __restrict__ positions,
   float3* __restrict__ sim_positions, int n_active_particles,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   int* __restrict__ n_bonds,
   unsigned char* __restrict__ broken_edges,
   int* __restrict__ accum_tri_counts,
   int label_component,
   float3* __restrict__ out_vertices, float3* __restrict__ out_normals,
   float3* __restrict__ out_uvs,
   int* __restrict__ out_tri_labels)
{  int block_id, idx, j, k, l, m;
   int label_id, part_id, point_id;
   int t_labels[4], t_part_ids[4], case_id, tri_counts;
   float3 t_vertices[4], s_vertices[N_MIDPOINTS];
   unsigned char *e, *s_idx, **b_idx, *vc_idx, *b;
   float I[9]={1,0,0, 0,1,0, 0,0,1}, *used_R;
   double t_abs_dets[4];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts = accum_tri_counts[idx];
   b = boundary_flags + 4*idx;

   if (tetras[idx*4] >= n_active_particles ||
       tetras[idx*4+1] >= n_active_particles ||
       tetras[idx*4+2] >= n_active_particles ||
       tetras[idx*4+3] >= n_active_particles)
      return;

   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_labels[j] = labels[part_id];
      t_part_ids[j] = part_id;
      t_abs_dets[j] = fabs(mat_det(A+9*part_id));
   }

   e = broken_edges+6*idx;
   case_id = 0;
   for (j=0; j < 6; j++)
      case_id |= e[j] << (5-j);

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = 3*tri_counts;
      // Loop through all triangle's vertices.
      for (l=0; l < 3; l++) {
         // For s_idx[1+...], we need skip one slot (s_idx[0]),
         //    which indicates a number of triangles.
         // surf_midpoints[s_idx[1+3*k+l]][1] gives a dominant vertex.
         point_id = surface_midpoints[s_idx[1+3*k+l]][1];
         part_id = t_part_ids[point_id];
         used_R = R+9*part_id;
         // if (n_bonds[part_id] < 2) {
         //    used_R = I;
         //    out_tri_labels[m+l] = -2;
         // }
         // else if (t_abs_dets[point_id] < DOUBLE_EPSILON) {
         //    used_R = I;
         //    out_tri_labels[m+l] = -1;
         // }
         // else {
         //    used_R = R+9*part_id;
         //    out_tri_labels[m+l] = label_id;
         // }
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         out_uvs[m+l] = s_vertices[s_idx[1+3*k+l]];
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[part_id], T1[part_id], used_R);
      }
      compute_normals(out_normals+m, out_vertices+m);
      tri_counts++;
   }

   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      if (b[j]) {
         // Loop through all boundary triangles.
         // b_idx[j] ~ a list of triangles of the face j.
         // b_idx[j][0] ~ a number of triangles of the face j.
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = 3*tri_counts;
            // Loop through all triangle's vertices.
            for (l=0; l < 3; l++) {
               point_id = surface_midpoints[b_idx[j][1+3*k+l]][1];
               part_id = t_part_ids[point_id];
               used_R = R+9*part_id;
               // if (n_bonds[part_id] < 2) {
               //    used_R = I;
               //    out_tri_labels[m+l] = -2;
               // }
               // else if (t_abs_dets[point_id] < DOUBLE_EPSILON) {
               //    used_R = I;
               //    out_tri_labels[m+l] = -1;
               // }
               // else {
               //    used_R = R+9*part_id;
               //    out_tri_labels[m+l] = label_id;
               // }
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               out_uvs[m+l] = s_vertices[b_idx[j][1+3*k+l]];
               // if (case_id != 0)
               //    out_tri_labels[m+l] = -1;
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[part_id], T1[part_id], used_R);
            }
            compute_normals(out_normals+m, out_vertices+m);
            tri_counts++;
         }
      }
   }
}

// Standard routine to extract fracture surface
// using per component transformation
__global__ void march_tetra_per_component_xform(float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   int* __restrict__ n_bonds,
   unsigned char* __restrict__ broken_edges,
   int* __restrict__ accum_tri_counts,
   int label_component,
   float3* __restrict__ out_vertices, float3* __restrict__ out_normals,
   float3* __restrict__ out_uvs,
   int* __restrict__ out_tri_labels)
{  int block_id, idx, j, k, l, m;
   int label_id, part_id;
   int t_labels[4], case_id, tri_counts;
   float3 t_vertices[4], s_vertices[N_MIDPOINTS];
   unsigned char *e, *s_idx, **b_idx, *vc_idx, *b;
   float *used_R;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts = accum_tri_counts[idx];
   b = boundary_flags + 4*idx;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_labels[j] = labels[part_id];
   }

   e = broken_edges+6*idx;
   case_id = 0;
   for (j=0; j < 6; j++)
      case_id |= e[j] << (5-j);

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = 3*tri_counts;
      // Loop through all triangle's vertices.
      for (l=0; l < 3; l++) {
         used_R = R+9*label_id;
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         out_uvs[m+l] = s_vertices[s_idx[1+3*k+l]];
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[label_id], T1[label_id], used_R);
      }
      compute_normals(out_normals+m, out_vertices+m);
      tri_counts++;
   }

   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      if (b[j]) {
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = 3*tri_counts;
            // Loop through all triangle's vertices.
            for (l=0; l < 3; l++) {
               used_R = R+9*label_id;
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               out_uvs[m+l] = s_vertices[b_idx[j][1+3*k+l]];
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[label_id], T1[label_id], used_R);
            }
            compute_normals(out_normals+m, out_vertices+m);
            tri_counts++;
         }
      }
   }
}

// Collect boundary tet faces and mark them in boundary_flags
__global__ void find_boundary(int* __restrict__ tetras,
   int* __restrict__ bounders, int n_tetras,
   unsigned char* __restrict__ boundary_flags)
{  int block_id, idx, j, k;
   int chk_list[12], chk_count, bnd_found, bnd0, bnd1, bnd2;
   unsigned char *b;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   b = boundary_flags + 4*idx;
   b[0] = b[1] = b[2] = b[3] = 0;

   chk_count = 0;
   // Loop through all vertices of the tet in the mesh file.
   //    If a boundary value of a vertex is -1, store the corresponding
   //    particle id (assumed to be on a boundary) in chk_list.
   // See also a mesh file format.
   for (j=0; j < 4; j++) {
      if (bounders[idx*4 + j] == -1)
         for (k=0; k < 4; k++)
            if (k != j)
               chk_list[chk_count++] = tetras[idx*4 + k];
   }
   if (chk_count == 0)
      return;
   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      // bnd0,1,2 are particle ids of a face.
      bnd0 = tetras[idx*4 + face_check[j][0]];
      bnd1 = tetras[idx*4 + face_check[j][1]];
      bnd2 = tetras[idx*4 + face_check[j][2]];
      bnd_found = 0;
      // If all 3 particle ids from a face we are about to generate are the
      //    same as ones from the mesh file, these particles are on a boundary.
      for (k=0; k < chk_count; k+=3)
         if ((bnd0==chk_list[k]||bnd0==chk_list[k+1]||bnd0==chk_list[k+2]) &&
             (bnd1==chk_list[k]||bnd1==chk_list[k+1]||bnd1==chk_list[k+2]) &&
             (bnd2==chk_list[k]||bnd2==chk_list[k+1]||bnd2==chk_list[k+2])) {
            bnd_found = 1;
            break;
         }
      b[j] = bnd_found;
   }
}

// Use geometric distance to determine edge breaking in the tetrahedron.
__global__ void breaking_edge_check(float3* __restrict__ positions,
   float3* __restrict__ sim_positions, int* __restrict__ tetras, int n_tetras,
   unsigned char* __restrict__ breaking_edges,
   unsigned char* __restrict__ broken_edges, float broken_distance)
{  int block_id, idx, j, k, l, part_id;
   float3 t_vertices[4], t_sim_vertices[4], *p, *s;
   unsigned char *e, *b, *ecp;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
   }

   e = breaking_edges+6*idx;
   b = broken_edges+6*idx;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;
   for (j=0; j < 6; j++) {
      e[j] = 0;
      if (b[j])  // if this edge was already broken, skip this edge.
         continue;
      k = ecp[2*j];
      l = ecp[2*j+1];
      e[j] = is_broken(0, 0, p[k], p[l], s[k], s[l], broken_distance);
   }
}

// Calculate the number of triangles based on the broken edges.
__global__ void tri_counts(unsigned char* __restrict__ boundary_flags,
   int n_tetras, unsigned char* __restrict__ broken_edges,
   int* __restrict__ tri_counts)
{  int block_id, idx, j, case_id;
   unsigned char *e, *s_idx, **b_idx, *b;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts[idx] = 0;
   b = boundary_flags + 4*idx;
   e = broken_edges+6*idx;
   case_id = 0;
   for (j=0; j < 6; j++)
      case_id |= e[j] << (5-j);

   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   tri_counts[idx] += s_idx[0];
   for (j=0; j < 4; j++) {
      if (b[j])
         tri_counts[idx] += b_idx[j][0];
   }
}

// Use geometric distance to determine edge breaks in the tetrahedron.
__global__ void broken_edge_check_tri_counts(float3* __restrict__ positions,
   float3* __restrict__ sim_positions, int n_active_particles,
   int* __restrict__ labels,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   unsigned char* __restrict__ broken_edges, float broken_distance,
   int* __restrict__ tri_counts)
{  int block_id, idx, j, k, l, part_id;
   int case_id, *t, t_labels[4];
   float3 t_vertices[4], t_sim_vertices[4], *p, *s;
   unsigned char *e, *ecp, *s_idx, **b_idx, *b;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts[idx] = 0;

   if (tetras[idx*4] >= n_active_particles ||
       tetras[idx*4+1] >= n_active_particles ||
       tetras[idx*4+2] >= n_active_particles ||
       tetras[idx*4+3] >= n_active_particles)
      return;

   b = boundary_flags + 4*idx;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
      t_labels[j] = labels[part_id];
   }

   e = broken_edges+6*idx;
   t = t_labels;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;
   case_id = 0;
   for (j=0; j < 6; j++) {
      k = ecp[2*j];
      l = ecp[2*j+1];
      e[j] |= is_broken(t[k], t[l], p[k], p[l], s[k], s[l], broken_distance);
      case_id |= e[j] << (5-j);
   }

   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   tri_counts[idx] += s_idx[0];
   for (j=0; j < 4; j++) {
      if (b[j])
         tri_counts[idx] += b_idx[j][0];
   }
}

// Use bond information to determine edge breaks in the tetrahedron.
__global__ void bond_broken_edge_tri_counts(float3* __restrict__ positions,
   float3* __restrict__ sim_positions, int n_active_particles,
   int* __restrict__ bondlist, int* __restrict__ n_bonds, int maxbonds,
   int* __restrict__ labels,
   unsigned char* __restrict__ boundary_flags,
   int* __restrict__ tetras, int n_tetras,
   unsigned char* __restrict__ broken_edges,
   int* __restrict__ tri_counts)
{  int block_id, idx, j, k, l, m, part_id, source, target;
   int case_id, *t, t_labels[4], t_part_ids[4];
   float3 t_vertices[4], t_sim_vertices[4], *p, *s;
   unsigned char *e, *ecp, *s_idx, **b_idx, *b;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts[idx] = 0;

   if (tetras[idx*4] >= n_active_particles ||
       tetras[idx*4+1] >= n_active_particles ||
       tetras[idx*4+2] >= n_active_particles ||
       tetras[idx*4+3] >= n_active_particles)
      return;

   b = boundary_flags + 4*idx;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_part_ids[j] = part_id;
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
      t_labels[j] = labels[part_id];
   }

   e = broken_edges+6*idx;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;
   case_id = 0;
   for (j=0; j < 6; j++) {
      k = ecp[2*j];
      l = ecp[2*j+1];
      if (e[j] == 0) {
         source = t_part_ids[k];
         target = t_part_ids[l];
         t = bondlist+source*maxbonds;
         for(m=0; m < n_bonds[source]; m++)
            if (t[m] == target)
               break;
         if (m == n_bonds[source])
            e[j] = 1;
      }
      case_id |= e[j] << (5-j);
   }

   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

   tri_counts[idx] += s_idx[0];
   for (j=0; j < 4; j++) {
      if (b[j])
         tri_counts[idx] += b_idx[j][0];
   }
}

// similar to march_tetra() except combining all routines into one function.
__global__ void march_tetra_per_particle_xform(float3* __restrict__ positions,
   float3* __restrict__ sim_positions,
   int* __restrict__ labels,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   // tetras contains a list of particle ids of tets.
   // bounders contains a list of opposite tet ids of a tet's vertex?
   int* __restrict__ tetras, int* __restrict__ bounders,
   int n_tetras, int* __restrict__ n_bonds,
   unsigned char* __restrict__ broken_edges, float broken_distance,
   int label_component,
   float3* __restrict__ out_vertices, int* __restrict__ tri_counts,
   int* __restrict__ out_tri_labels,
   int tris_per_tet)
{  int block_id, idx, j, k, l, m, bnd_found;
   int label_id, part_id, point_id;
   int t_labels[4], t_part_ids[4], case_id, *t, bnd0, bnd1, bnd2;
   int chk_list[12], chk_count;
   float3 t_vertices[4], t_sim_vertices[4], s_vertices[N_MIDPOINTS], *p, *s;
   unsigned char *e, *ecp, *s_idx, **b_idx, *vc_idx;
   float I[9]={1,0,0, 0,1,0, 0,0,1}, *used_R;
   double t_abs_dets[4];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_tetras)
      return;

   tri_counts[idx] = 0;
   for (j=0; j < 4; j++) {
      part_id = tetras[idx*4 + j];
      t_vertices[j] = positions[part_id];
      t_sim_vertices[j] = sim_positions[part_id];
      t_labels[j] = labels[part_id];
      t_part_ids[j] = part_id;
      t_abs_dets[j] = fabs(mat_det(A+9*part_id));
   }

   e = broken_edges+6*idx;
   t = t_labels;
   p = t_vertices;
   s = t_sim_vertices;
   ecp = edge_connected_points;
   case_id = 0;
   for (j=0; j < 6; j++) {
      k = ecp[2*j];
      l = ecp[2*j+1];
      e[j] |= is_broken(t[k], t[l], p[k], p[l], s[k], s[l], broken_distance);
      case_id |= e[j] << (5-j);
   }

   vc_idx = v_compute[case_id];
   s_idx  = inner_surface[case_id];
   b_idx  = boundary_surface[case_id];

#if 0   // use equal weights!
   // give all with equal weights.
   float surf_weights[88][4] = {{1.}, {1.}, {1.}, {1.}, {0.}};
   for (j=0; j < sizeof(surface_midpoints)/sizeof(surface_midpoints[0]);
        j++) {
      for (k=0; k < surface_midpoints[j][0]; k++)
         surf_weights[j][k] = 1./surface_midpoints[j][0];
   }
   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surf_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }
#else
   // compute all necessary vertices and mid-points.
   for (j=1; j <= vc_idx[0]; j++) {
      m = vc_idx[j];
      s_vertices[m].x = s_vertices[m].y = s_vertices[m].z = 0;
      for (k=1; k <= surface_midpoints[m][0]; k++)
         s_vertices[m] += surface_weights[m][k-1] *
                             t_vertices[surface_midpoints[m][k]];
   }
#endif

   // Loop through all triangles, one at a time.
   for (k=0; k < s_idx[0]; k++) {
      label_id = t_labels[surface_midpoints[s_idx[1+3*k]][1]];
      m = idx*tris_per_tet*3 + 3*tri_counts[idx];
      // Loop through all triangle's vertices.
      for (l=0; l < 3; l++) {
         // For s_idx[1+...], we need skip one slot (s_idx[0]),
         //    which indicates a number of triangles.
         // surf_midpoints[s_idx[1+3*k+l]][1] gives a dominant vertex.
         point_id = surface_midpoints[s_idx[1+3*k+l]][1];
         part_id = t_part_ids[point_id];
         used_R = R+9*part_id;
         // if (n_bonds[part_id] < 2) {
         //    used_R = I;
         //    out_tri_labels[m+l] = -2;
         // }
         // else if (t_abs_dets[point_id] < DOUBLE_EPSILON) {
         //    used_R = I;
         //    out_tri_labels[m+l] = -1;
         // }
         // else {
         //    used_R = R+9*part_id;
         //    out_tri_labels[m+l] = label_id;
         // }
         if (label_component)
            out_tri_labels[m+l] = label_id;
         else
            out_tri_labels[m+l] = 1;  // interior face
         vertex_xform(out_vertices+m+l, s_vertices[s_idx[1+3*k+l]],
            T0[part_id], T1[part_id], used_R);
      }
      tri_counts[idx]++;
   }

   chk_count = 0;
   // Loop through all vertices of the tet in the mesh file.
   //    If a boundary value of a vertex is -1, store the corresponding
   //    particle id (assumed to be on a boundary) in chk_list.
   // See also a mesh file format.
   for (j=0; j < 4; j++) {
      if (bounders[idx*4 + j] == -1)
         for (k=0; k < 4; k++)
            if (k != j)
               chk_list[chk_count++] = tetras[idx*4 + k];
   }
   if (chk_count == 0)
      return;
   // Loop through all 4 faces of a tet.
   for (j=0; j < 4; j++) {
      // bnd0,1,2 are particle ids of a face.
      bnd0 = tetras[idx*4 + face_check[j][0]];
      bnd1 = tetras[idx*4 + face_check[j][1]];
      bnd2 = tetras[idx*4 + face_check[j][2]];
      bnd_found = 0;
      // If all 3 particle ids from a face we are about to generate are the
      //    same as ones from the mesh file, these particles are on a boundary.
      for (k=0; k < chk_count; k+=3)
         if ((bnd0==chk_list[k]||bnd0==chk_list[k+1]||bnd0==chk_list[k+2]) &&
             (bnd1==chk_list[k]||bnd1==chk_list[k+1]||bnd1==chk_list[k+2]) &&
             (bnd2==chk_list[k]||bnd2==chk_list[k+1]||bnd2==chk_list[k+2])) {
            bnd_found = 1;
            break;
         }
      // Generating boundary triangles.
      if (bnd_found == 1) {
         // Loop through all boundary triangles.
         // b_idx[j] ~ a list of triangles of the face j.
         // b_idx[j][0] ~ a number of triangles of the face j.
         for (k=0; k < b_idx[j][0]; k++) {
            label_id = t_labels[surface_midpoints[b_idx[j][1+3*k]][1]];
            m = idx*tris_per_tet*3 + 3*tri_counts[idx];
            // Loop through all triangle's vertices.
            for (l=0; l < 3; l++) {
               point_id = surface_midpoints[b_idx[j][1+3*k+l]][1];
               part_id = t_part_ids[point_id];
               used_R = R+9*part_id;
               // if (n_bonds[part_id] < 2) {
               //    used_R = I;
               //    out_tri_labels[m+l] = -2;
               // }
               // else if (t_abs_dets[point_id] < DOUBLE_EPSILON) {
               //    used_R = I;
               //    out_tri_labels[m+l] = -1;
               // }
               // else {
               //    used_R = R+9*part_id;
               //    out_tri_labels[m+l] = label_id;
               // }
               if (label_component)
                  out_tri_labels[m+l] = label_id;
               else
                  out_tri_labels[m+l] = 0;   // exterior face
               vertex_xform(out_vertices+m+l, s_vertices[b_idx[j][1+3*k+l]],
                  T0[part_id], T1[part_id], used_R);
            }
            tri_counts[idx]++;
         }
      }
   }
}

__global__ void pack_vertices_labels(float3* __restrict__ vertices,
   int* __restrict__ tri_counts, int* __restrict__ tri_labels,
   float3* __restrict__ packed_vertices, float3* __restrict__ packed_normals,
   int* __restrict__ packed_labels, int tris_per_cube, int max_index)
{  int i, block_id, idx, cnt, k, l;
   float3 e0, e1, n;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= max_index)
      return;
   cnt = 0;
   for (i=0; i < idx; i++)
      cnt += tri_counts[i];
   for (i=0; i < tri_counts[idx]; i++) {
      k = 3*cnt + 3*i;  l = idx*tris_per_cube*3 + 3*i;
      packed_vertices[k]   = vertices[l];
      packed_vertices[k+1] = vertices[l+1];
      packed_vertices[k+2] = vertices[l+2];
      e0 = packed_vertices[k]   - packed_vertices[k+1];
      e1 = packed_vertices[k+2] - packed_vertices[k+1];
      e0 = normalize(e0);
      e1 = normalize(e1);
      n  = cross(e0, e1);
      packed_normals[k]   = n;
      packed_normals[k+1] = n;
      packed_normals[k+2] = n;
      packed_labels[k]    = tri_labels[l];
      packed_labels[k+1]  = tri_labels[l+1];
      packed_labels[k+2]  = tri_labels[l+2];
   }
}

__global__ void pack_vertices(float3* __restrict__ vertices,
   int* __restrict__ tri_counts, float3* __restrict__ packed_vertices,
   float3* __restrict__ packed_normals, int tris_per_cube, int max_index)
{  int i, block_id, idx, cnt, k, l;
   float3 e0, e1, n;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= max_index)
      return;
   cnt = 0;
   for (i=0; i < idx; i++)
      cnt += tri_counts[i];
   for (i=0; i < tri_counts[idx]; i++) {
      k = 3*cnt + 3*i;  l = idx*tris_per_cube*3 + 3*i;
      packed_vertices[k]   = vertices[l];
      packed_vertices[k+1] = vertices[l+1];
      packed_vertices[k+2] = vertices[l+2];
      e0 = packed_vertices[k]   - packed_vertices[k+1];
      e1 = packed_vertices[k+2] - packed_vertices[k+1];
      e0 = normalize(e0);
      e1 = normalize(e1);
      n  = cross(e0, e1);
      packed_normals[k]   = n;
      packed_normals[k+1] = n;
      packed_normals[k+2] = n;
   }
}

__global__ void label_particles(int* __restrict__ bondlist,
   int* __restrict__ n_bonds, int n_particles, int maxbonds,
   int* __restrict__ labels, int* __restrict__ n_updates)
{  int i, block_id, idx, min_label;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   min_label = labels[idx];
   for (i=0; i < n_bonds[idx]; i++) {
      if (min_label > labels[bondlist[idx*maxbonds+i]])
         min_label = labels[bondlist[idx*maxbonds+i]];
   }
   if (min_label != labels[idx]) {
      atomicAdd(n_updates, 1);
      labels[idx] = min_label;
   }
}

__global__ void mark_labels(int n_particles, int* __restrict__ labels,
   unsigned char* __restrict__ marked_labels)
{  int i, block_id, idx;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   marked_labels[idx] = 0;
   for (i=0; i < n_particles; i++) {
      if (labels[i] == idx) {
         marked_labels[idx] = 1;
         break;
      }
   }
}

__global__ void renum_labels(int n_particles, int* __restrict__ labels,
   unsigned char* __restrict__ marked_labels,
   int* __restrict__ out_labels)
{  int i, block_id, idx, num;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   num = 0;
   for (i=0; i < labels[idx]; i++) {
      if (marked_labels[i] == 0)
         num++;
   }
   out_labels[idx] = labels[idx] - num;
}

/* Taken from http://www.ngs.noaa.gov/gps-toolbox/sp3intrp/svdfit.c */
/* Slightly modified versions of routines from
 * Press, William H., Brian P. Flannery, Saul A Teukolsky and
 *   William T. Vetterling, 1986, "Numerical Recipes: The Art of
 *   Scientific Computing" (Fortran), Cambrigde University Press.
 *
 * svd  on pp. 60-64.
 */
__device__ int svd(float *A, float *W, float *V)
{
    /*
       Give a matrix A, with physical dimensions M by N, this routine computes its singular value decomposition, A = U * W * transpose V. The matrix U replaces A on output. The diagonal matrix of singular values, W, is output as a vector W. The matrix V (not the transpose of V) is output as
       V. M must be greater or equal to N. If it is smaller then A should be filled up to square with zero rows.
    */

    unsigned int M=3, N=3;
    double rv1[3];

    /* Householder reduction to bidiagonal form. */
    int NM;
    double C, F, G = 0.0, H, S, X, Y, Z, Scale = 0.0, ANorm = 0.0, tmp;
    int flag, i, its, j, jj, k, l;

    for( i = 0; i < N; ++i ) {
        l = i + 1;
        rv1[i] = Scale * G;
        G = 0.0;
        S = 0.0;
        Scale = 0.0;
        if( i < M ) {
            for( k = i; k < M; ++k ) {
                Scale = Scale + fabs( A[3*k+i] );
            }
            if( Scale != 0.0 ) {
                for( k = i; k < M; ++k ) {
                    A[3*k+i] = A[3*k+i] / Scale;
                    S = S + A[3*k+i] * A[3*k+i];
                }
                F = A[3*i+i];
                G = sqrt(S);
                if( F > 0.0 ) {
                    G = -G;
                }
                H = F * G - S;
                A[3*i+i] = F - G;
                if( i != (N-1) ) {
                    for( j = l; j < N; ++j ) {
                        S = 0.0;
                        for( k = i; k < M; ++k ) {
                            S = S + A[3*k+i] * A[3*k+j];
                        }
                        F = S / H;
                        for( k = i; k < M; ++k ) {
                            A[3*k+j] = A[3*k+j] + F * A[3*k+i];
                        }
                    }
                }
                for( k = i; k < M; ++k ) {
                    A[3*k+i] = Scale * A[3*k+i];
                }
            }
        }

        W[i] = Scale * G;
        G = 0.0;
        S = 0.0;
        Scale = 0.0;
        if( (i < M) && (i != (N-1)) ) {
            for( k = l; k < N; ++k ) {
                Scale = Scale + fabs( A[3*i+k] );
            }
            if( Scale != 0.0 ) {
                for( k = l; k < N; ++k ) {
                    A[3*i+k] = A[3*i+k] / Scale;
                    S = S + A[3*i+k] * A[3*i+k];
                }
                F = A[3*i+l];
                G = sqrt(S);
                if( F > 0.0 ) {
                    G = -G;
                }
                H = F * G - S;
                A[3*i+l] = F - G;
                for( k = l; k < N; ++k ) {
                    rv1[k] = A[3*i+k] / H;
                }
                if( i != (M-1) ) {
                    for( j = l; j < M; ++j ) {
                        S = 0.0;
                        for( k = l; k < N; ++k ) {
                            S = S + A[3*j+k] * A[3*i+k];
                        }
                        for( k = l; k < N; ++k ) {
                            A[3*j+k] = A[3*j+k] + S * rv1[k];
                        }
                    }
                }
                for( k = l; k < N; ++k ) {
                    A[3*i+k] = Scale * A[3*i+k];
                }
            }
        }
        tmp = fabs( W[i] ) + fabs( rv1[i] );
        if( tmp > ANorm )
            ANorm = tmp;
    }

    /* Accumulation of right-hand transformations. */
    for( i = N-1; i >= 0; --i ) {
        if( i < (N-1) ) {
            if( G != 0.0 ) {
                for( j = l; j < N; ++j ) {
                    V[3*j+i] = (A[3*i+j] / A[3*i+l]) / G;
                }
                for( j = l; j < N; ++j ) {
                    S = 0.0;
                    for( k = l; k < N; ++k ) {
                        S = S + A[3*i+k] * V[3*k+j];
                    }
                    for( k = l; k < N; ++k ) {
                        V[3*k+j] = V[3*k+j] + S * V[3*k+i];
                    }
                }
            }
            for( j = l; j < N; ++j ) {
                V[3*i+j] = 0.0;
                V[3*j+i] = 0.0;
            }
        }
        V[3*i+i] = 1.0;
        G = rv1[i];
        l = i;
    }

    /* Accumulation of left-hand transformations. */
    for( i = N-1; i >= 0; --i ) {
        l = i + 1;
        G = W[i];
        if( i < (N-1) ) {
            for( j = l; j < N; ++j ) {
                A[3*i+j] = 0.0;
            }
        }
        if( G != 0.0 ) {
            G = 1.0 / G;
            if( i != (N-1) ) {
                for( j = l; j < N; ++j ) {
                    S = 0.0;
                    for( k = l; k < M; ++k ) {
                        S = S + A[3*k+i] * A[3*k+j];
                    }
                    F = (S / A[3*i+i]) * G;
                    for( k = i; k < M; ++k ) {
                        A[3*k+j] = A[3*k+j] + F * A[3*k+i];
                    }
                }
            }
            for( j = i; j < M; ++j ) {
                A[3*j+i] = A[3*j+i] * G;
            }
        } else {
            for( j = i; j < M; ++j ) {
                A[3*j+i] = 0.0;
            }
        }
        A[3*i+i] = A[3*i+i] + 1.0;
    }

    /* Diagonalization of the bidiagonal form.
       Loop over singular values. */
    for( k = (N-1); k >= 0; --k ) {
        /* Loop over allowed iterations. */
        for( its = 1; its <= 300; ++its ) {
            /* Test for splitting.
               Note that rv1[0] is always zero. */
            flag = true;
            for( l = k; l >= 0; --l ) {
                NM = l - 1;
                if( (fabs(rv1[l]) + ANorm) == ANorm ) {
                    flag = false;
                    break;
                } else if( (fabs(W[NM]) + ANorm) == ANorm ) {
                    break;
                }
            }

            /* Cancellation of rv1[l], if l > 0; */
            if( flag ) {
                C = 0.0;
                S = 1.0;
                for( i = l; i <= k; ++i ) {
                    F = S * rv1[i];
                    if( (fabs(F) + ANorm) != ANorm ) {
                        G = W[i];
                        H = sqrt( F * F + G * G );
                        W[i] = H;
                        H = 1.0 / H;
                        C = ( G * H );
                        S = -( F * H );
                        for( j = 0; j < M; ++j ) {
                            Y = A[3*j+NM];
                            Z = A[3*j+i];
                            A[3*j+NM] = (Y * C) + (Z * S);
                            A[3*j+i] = -(Y * S) + (Z * C);
                        }
                    }
                }
            }
            Z = W[k];
            /* Convergence. */
            if( l == k ) {
                /* Singular value is made nonnegative. */
                if( Z < 0.0 ) {
                    W[k] = -Z;
                    for( j = 0; j < N; ++j ) {
                        V[3*j+k] = -V[3*j+k];
                    }
                }
                break;
            }

            if( its >= 300 )
                return 0;

            X = W[l];
            NM = k - 1;
            Y = W[NM];
            G = rv1[NM];
            H = rv1[k];
            F = ((Y-Z)*(Y+Z) + (G-H)*(G+H)) / (2.0*H*Y);
            G = sqrt( F * F + 1.0 );
            tmp = G;
            if( F < 0.0 )
                tmp = -tmp;
            F = ((X-Z)*(X+Z) + H*((Y/(F+tmp))-H)) / X;

            /* Next QR transformation. */
            C = 1.0;
            S = 1.0;
            for( j = l; j <= NM; ++j ) {
                i = j + 1;
                G = rv1[i];
                Y = W[i];
                H = S * G;
                G = C * G;
                Z = sqrt( F * F + H * H );
                rv1[j] = Z;
                C = F / Z;
                S = H / Z;
                F = (X * C) + (G * S);
                G = -(X * S) + (G * C);
                H = Y * S;
                Y = Y * C;
                for( jj = 0; jj < N; ++jj ) {
                    X = V[3*jj+j];
                    Z = V[3*jj+i];
                    V[3*jj+j] = (X * C) + (Z * S);
                    V[3*jj+i] = -(X * S) + (Z * C);
                }
                Z = sqrt( F * F + H * H );
                W[j] = Z;

                /* Rotation can be arbitrary if Z = 0. */
                if( Z != 0.0 ) {
                    Z = 1.0 / Z;
                    C = F * Z;
                    S = H * Z;
                }
                F = (C * G) + (S * Y);
                X = -(S * G) + (C * Y);
                for( jj = 0; jj < M; ++jj ) {
                    Y = A[3*jj+j];
                    Z = A[3*jj+i];
                    A[3*jj+j] = (Y * C) + (Z * S);
                    A[3*jj+i] = -(Y * S) + (Z * C);
                }
            }
            rv1[l] = 0.0;
            rv1[k] = F;
            W[k] = X;
        }
    }

    return 1;
}

__device__ void add_inner_product(float *U, float3 Po, float3 Qo)
{
   U[0] += Po.x * Qo.x;
   U[1] += Po.x * Qo.y;
   U[2] += Po.x * Qo.z;
   U[3] += Po.y * Qo.x;
   U[4] += Po.y * Qo.y;
   U[5] += Po.y * Qo.z;
   U[6] += Po.z * Qo.x;
   U[7] += Po.z * Qo.y;
   U[8] += Po.z * Qo.z;
}

__global__ void compute_per_particle_procrustes(
   float3* __restrict__ org_vertices,
   float3* __restrict__ sim_vertices, int n_particles,
   int* __restrict__ bondlist, int* __restrict__ n_bonds, int maxbonds,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   float3* __restrict__  random_scale)
{  int i, block_id, idx;
   float3 Cp, Cq, Po, Qo;
   float3 p0, s0;
   float *M, *N, U[9], S[3], V[9];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   p0 = org_vertices[idx];
   s0 = sim_vertices[idx];

   // T0[idx] = make_float3(0,0,0);
   // T1[idx] = make_float3(0,0,0);
   // R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
   //    R[idx*9+7] = 0;
   // R[idx*9]   = 1.0;
   // R[idx*9+4] = 1.0;
   // R[idx*9+8] = 1.0;
   // return;

   float limit = 0.1;
   float scale = 0.98;
   if (n_bonds[idx] == 0) {
      T0[idx] = p0;
      T1[idx] = s0;
      // if (R[idx*9] > limit || R[idx*9+4] > limit || R[idx*9+8] > limit) {
      //   R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
      //      R[idx*9+7] = 0;
      //   R[idx*9] *= scale;
      //   R[idx*9+4] *= scale;
      //   R[idx*9+8] *= scale;
      // }
      // R[idx*9] = random_scale[idx].x;
      // R[idx*9+4] = random_scale[idx].y;
      // R[idx*9+8] = random_scale[idx].z;
      return;
   }
   T0[idx] = Cp = p0;
   T1[idx] = Cq = s0;
   if (n_bonds[idx] < 3) {
      // R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
      //    R[idx*9+7] = 0;
      // R[idx*9] = R[idx*9+4] = R[idx*9+8] = 1;
      return;
   }
   M = R+idx*9;
   U[0] = U[1] = U[2] = U[3] = U[4] = U[5] = U[6] = U[7] = U[8] = 0;
   for (i=0; i < n_bonds[idx]; i++) {
      Po = org_vertices[bondlist[idx*maxbonds+i]] - Cp;
      Qo = sim_vertices[bondlist[idx*maxbonds+i]] - Cq;
      add_inner_product(U, Po, Qo);
   }
   N = A+idx*9;
   N[0] = U[0]; N[1] = U[1]; N[2] = U[2];
   N[3] = U[3]; N[4] = U[4]; N[5] = U[5];
   N[6] = U[6]; N[7] = U[7]; N[8] = U[8];
   if (!svd(U, S, V)) {
      // R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
      //    R[idx*9+7] = 0;
      // R[idx*9] = R[idx*9+4] = R[idx*9+8] = 1;
      return;
   }
   // M = U x V.T
   M[0] = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];
   M[1] = U[0]*V[3] + U[1]*V[4] + U[2]*V[5];
   M[2] = U[0]*V[6] + U[1]*V[7] + U[2]*V[8];
   M[3] = U[3]*V[0] + U[4]*V[1] + U[5]*V[2];
   M[4] = U[3]*V[3] + U[4]*V[4] + U[5]*V[5];
   M[5] = U[3]*V[6] + U[4]*V[7] + U[5]*V[8];
   M[6] = U[6]*V[0] + U[7]*V[1] + U[8]*V[2];
   M[7] = U[6]*V[3] + U[7]*V[4] + U[8]*V[5];
   M[8] = U[6]*V[6] + U[7]*V[7] + U[8]*V[8];
}

__global__ void compute_per_particle_procrustes_cem(
   float3* __restrict__ org_vertices,
   float3* __restrict__ sim_vertices, int n_particles,
   int* __restrict__ bondlist, int* __restrict__ n_bonds, int maxbonds,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   float3* __restrict__  random_scale)
{  int i, block_id, idx, cnt;
   float3 Cp, Cq, Po, Qo;
   float3 p0, s0;
   float *M, *N, U[9], S[3], V[9];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   p0 = org_vertices[idx];
   s0 = sim_vertices[idx];

   float limit = 0.1;
   float scale = 0.98;
   if (n_bonds[idx] == 0) {
      T0[idx] = p0;
      T1[idx] = s0;
      R[idx*9] *= 0.1;
      R[idx*9+4] = 0.1;
      R[idx*9+8] = 0.1;
      return;
   }
   T0[idx] = Cp = p0;
   T1[idx] = Cq = s0;
   if (n_bonds[idx] < 3)
      return;
   M = R+idx*9;
   U[0] = U[1] = U[2] = U[3] = U[4] = U[5] = U[6] = U[7] = U[8] = 0;
   for (i=cnt=0; i < n_bonds[idx]; i++) {
      Po = org_vertices[bondlist[idx*maxbonds+i]] - Cp;
      Qo = sim_vertices[bondlist[idx*maxbonds+i]] - Cq;
      if (sqrt(dot(Qo, Qo)) > 0.005*0.005)
         continue;
      add_inner_product(U, Po, Qo);
      cnt++;
   }
   if (cnt < 5) {
      n_bonds[idx] = 0;
      R[idx*9] *= 0.1;
      R[idx*9+4] = 0.1;
      R[idx*9+8] = 0.1;
      return;
   }
   N = A+idx*9;
   N[0] = U[0]; N[1] = U[1]; N[2] = U[2];
   N[3] = U[3]; N[4] = U[4]; N[5] = U[5];
   N[6] = U[6]; N[7] = U[7]; N[8] = U[8];
   if (!svd(U, S, V))
      return;
   // M = U x V.T
   M[0] = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];
   M[1] = U[0]*V[3] + U[1]*V[4] + U[2]*V[5];
   M[2] = U[0]*V[6] + U[1]*V[7] + U[2]*V[8];
   M[3] = U[3]*V[0] + U[4]*V[1] + U[5]*V[2];
   M[4] = U[3]*V[3] + U[4]*V[4] + U[5]*V[5];
   M[5] = U[3]*V[6] + U[4]*V[7] + U[5]*V[8];
   M[6] = U[6]*V[0] + U[7]*V[1] + U[8]*V[2];
   M[7] = U[6]*V[3] + U[7]*V[4] + U[8]*V[5];
   M[8] = U[6]*V[6] + U[7]*V[7] + U[8]*V[8];
}

__global__ void compute_per_component_procrustes(
   float3* __restrict__ org_vertices,
   float3* __restrict__ sim_vertices, int n_particles,
   int* __restrict__ labels, int max_labels,
   int* __restrict__ bondlist, int* __restrict__ n_bonds, int maxbonds,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   float3* __restrict__  random_scale)
{  int i, block_id, idx, cnt;
   float3 Cp, Cq, Po, Qo;
   float3 p0, s0;
   float *M, *N, U[9], S[3], V[9];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= max_labels)
      return;

   p0 = org_vertices[idx];
   s0 = sim_vertices[idx];
   if (n_bonds[idx] == 0) {
      T0[idx] = p0;
      T1[idx] = s0;
      //if (R[idx*9] > 0.5 || R[idx*9+4] > 0.5 || R[idx*9+8] > 0.5) {
      //   R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
      //      R[idx*9+7] = 0;
      //   R[idx*9] *= 0.5;
      //   R[idx*9+4] *= 0.5;
      //   R[idx*9+8] *= 0.5;
      //}
      // R[idx*9] = random_scale[idx].x;
      // R[idx*9+4] = random_scale[idx].y;
      // R[idx*9+8] = random_scale[idx].z;
      return;
   }
   Cp = p0;
   Cq = s0;
   cnt = 1;
   for (i=0; i < max_labels; i++) {
      if (labels[i] != idx)
         continue;
      Cp += org_vertices[i];
      Cq += sim_vertices[i];
      cnt++;
   }
   if (cnt > 0) {
      Cp *= (1./cnt);
      Cq *= (1./cnt);
   }
   T0[idx] = Cp;
   T1[idx] = Cq;
   if (cnt < 3)
      return;

   M = R+idx*9;
   U[0] = U[1] = U[2] = U[3] = U[4] = U[5] = U[6] = U[7] = U[8] = 0;
   add_inner_product(U, p0-Cp, s0-Cq);
   for (i=0; i < max_labels; i++) {
      if (labels[i] != idx)
         continue;
      Po = org_vertices[i] - Cp;
      Qo = sim_vertices[i] - Cq;
      add_inner_product(U, Po, Qo);
   }
   N = A+idx*9;
   N[0] = U[0]; N[1] = U[1]; N[2] = U[2];
   N[3] = U[3]; N[4] = U[4]; N[5] = U[5];
   N[6] = U[6]; N[7] = U[7]; N[8] = U[8];
   if (!svd(U, S, V))
      return;

   // M = U x V.T
   M[0] = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];
   M[1] = U[0]*V[3] + U[1]*V[4] + U[2]*V[5];
   M[2] = U[0]*V[6] + U[1]*V[7] + U[2]*V[8];
   M[3] = U[3]*V[0] + U[4]*V[1] + U[5]*V[2];
   M[4] = U[3]*V[3] + U[4]*V[4] + U[5]*V[5];
   M[5] = U[3]*V[6] + U[4]*V[7] + U[5]*V[8];
   M[6] = U[6]*V[0] + U[7]*V[1] + U[8]*V[2];
   M[7] = U[6]*V[3] + U[7]*V[4] + U[8]*V[5];
   M[8] = U[6]*V[6] + U[7]*V[7] + U[8]*V[8];
}

__global__ void compute_per_particle_procrustes_about_centroid(
   float3* __restrict__ org_vertices,
   float3* __restrict__ sim_vertices, int n_particles,
   int* __restrict__ bondlist, int* __restrict__ n_bonds, int maxbonds,
   float3* __restrict__ T0, float3* __restrict__ T1,
   float* __restrict__ R, float* __restrict__ A,
   float3* __restrict__  random_scale)
{  int i, block_id, idx;
   float3 Cp, Cq, Po, Qo;
   float3 p0, s0;
   float *M, *N, U[9], S[3], V[9];

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   p0 = org_vertices[idx];
   s0 = sim_vertices[idx];
   if (n_bonds[idx] == 0) {
      T0[idx] = p0;
      T1[idx] = s0;
      R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
         R[idx*9+7] = 0;
      R[idx*9] = R[idx*9+4] = R[idx*9+8] = 1;
      // R[idx*9] = random_scale[idx].x;
      // R[idx*9+4] = random_scale[idx].y;
      // R[idx*9+8] = random_scale[idx].z;
      return;
   }
   Cp = p0;
   Cq = s0;
   for (i=0; i < n_bonds[idx]; i++) {
      Cp += org_vertices[bondlist[idx*maxbonds+i]];
      Cq += sim_vertices[bondlist[idx*maxbonds+i]];
   }
   if (n_bonds[idx] > 0) {
      Cp *= (1./(n_bonds[idx]+1));
      Cq *= (1./(n_bonds[idx]+1));
   }
   T0[idx] = Cp;
   T1[idx] = Cq;
   if (n_bonds[idx]+1 < 3) {
      R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
         R[idx*9+7] = 0;
      R[idx*9] = R[idx*9+4] = R[idx*9+8] = 1;
      return;
   }
   M = R+idx*9;
   U[0] = U[1] = U[2] = U[3] = U[4] = U[5] = U[6] = U[7] = U[8] = 0;
   add_inner_product(U, p0-Cp, s0-Cq);
   for (i=0; i < n_bonds[idx]; i++) {
      Po = org_vertices[bondlist[idx*maxbonds+i]] - Cp;
      Qo = sim_vertices[bondlist[idx*maxbonds+i]] - Cq;
      add_inner_product(U, Po, Qo);
   }
   N = A+idx*9;
   N[0] = U[0]; N[1] = U[1]; N[2] = U[2];
   N[3] = U[3]; N[4] = U[4]; N[5] = U[5];
   N[6] = U[6]; N[7] = U[7]; N[8] = U[8];
   if (!svd(U, S, V)) {
      R[idx*9+1] = R[idx*9+2] = R[idx*9+3] = R[idx*9+5] = R[idx*9+6] =
         R[idx*9+7] = 0;
      R[idx*9] = R[idx*9+4] = R[idx*9+8] = 1;
      return;
   }
   // M = U x V.T
   M[0] = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];
   M[1] = U[0]*V[3] + U[1]*V[4] + U[2]*V[5];
   M[2] = U[0]*V[6] + U[1]*V[7] + U[2]*V[8];
   M[3] = U[3]*V[0] + U[4]*V[1] + U[5]*V[2];
   M[4] = U[3]*V[3] + U[4]*V[4] + U[5]*V[5];
   M[5] = U[3]*V[6] + U[4]*V[7] + U[5]*V[8];
   M[6] = U[6]*V[0] + U[7]*V[1] + U[8]*V[2];
   M[7] = U[6]*V[3] + U[7]*V[4] + U[8]*V[5];
   M[8] = U[6]*V[6] + U[7]*V[7] + U[8]*V[8];
}

// Compute FTLEs.
__global__ void compute_FTLE(float3* __restrict__ positions,
   float3* __restrict__ next_positions, int n_particles,
   int* __restrict__ bondlist, int* __restrict__ n_bonds,
   int* __restrict__ accum_n_bonds, int maxbonds,
   float inv_ftle_tau,
   float3* __restrict__ locations, float* __restrict__ ftles)
{  int block_id, idx, j, k, l, m;
   float3 diff0, diff1;
   float d0, d1;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_particles)
      return;

   m = accum_n_bonds[idx];
   for (j=0; j < n_bonds[idx]; j++) {
      k = bondlist[idx*maxbonds+j];
      diff0 = positions[idx] - positions[k];
      diff1 = next_positions[idx] - next_positions[k];
      d0 = sqrt(dot(diff0, diff0));
      d1 = sqrt(dot(diff1, diff1));
      locations[m] = 0.5*(positions[idx]+positions[k]);
      ftles[m] = inv_ftle_tau*log(d1/d0);
      m++;
   }
}

__global__ void remove_duplicates1(float3* __restrict__ vertices,
   int* __restrict__ faces, int* __restrict__ out_faces,
   int n_vertices)
{  int i, block_id, idx;
   float3 d, v;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_vertices)   // idx refers to vertex id.
      return;

   v = vertices[faces[idx]];
   for (i=0; i < idx; i++) {
       d = vertices[faces[i]] - v;
       if (dot(d, d) < EPSILON && out_faces[idx] > faces[i]) {
          out_faces[idx] = faces[i];
          break;
       }
   }
}

__global__ void remove_duplicates(float3* __restrict__ vertices,
   int* __restrict__ faces, int n_slices,
   int* __restrict__ out_faces,
   int n_vertices, int *actual_n_vertices)
{  int i, block_id, idx, min_vert_id, vert_id;
   int slice_id, per_slice, end_slice;
   float3 d, v;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_slices*n_vertices)   // idx refers to vertex id.
      return;

   slice_id = idx % n_slices;
   vert_id = idx / n_slices;
   per_slice = n_vertices / n_slices;
   min_vert_id = faces[vert_id];
   v = vertices[faces[min_vert_id]];
   if (slice_id == n_slices-1)
      end_slice = n_vertices;
   else
      end_slice = (slice_id+1)*per_slice;
   for (i=slice_id*per_slice; i < end_slice; i++) {
      d = vertices[faces[i]] - v;
      if (dot(d, d) < EPSILON && min_vert_id > faces[i])
         min_vert_id = faces[i];
   }
   atomicMin(out_faces+vert_id, min_vert_id);
   __syncthreads();
   // if (min_vert_id == idx) {
   //    vert_id = atomicAdd(actual_n_vertices, 1);
   //    out_faces[idx] = vert_id;
   //    out_vertices[vert_id] = vertices[idx];
   // }
   // else
   //    out_faces[idx] = vert_id;
}

__global__ void count_vertices(int* __restrict__ faces,
   int* __restrict__ counts, int n_vertices, int *actual_n_vertices)
{  int i, block_id, idx, cnt;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_vertices)
      return;

   cnt = 0;
   for (i=0; i < n_vertices; i++) {
      if (faces[i] == idx) {
         cnt++;
         break;
      }
   }
   counts[idx] = cnt;
   if (cnt > 0)
      atomicAdd(actual_n_vertices, 1);
}

__global__ void accum_zero_counts(int* __restrict__ counts,
   int* __restrict__ accum_zero_counts, int n_vertices)
{  int i, block_id, idx, cnt;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_vertices)
      return;

   cnt = 0;
   for (i=0; i < idx; i++) {
      if (counts[i] == 0)
         cnt++;
   }
   accum_zero_counts[idx] = cnt;
}

__global__ void remove_vertices(float3* __restrict__ vertices,
   float3* __restrict__ out_vertices, int* __restrict__ faces,
   int* __restrict__ counts, int* __restrict__ accum_zero_counts,
   int* __restrict__ out_faces, int n_vertices)
{  int block_id, idx;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_vertices)
      return;

   out_faces[idx] = faces[idx] - accum_zero_counts[idx];
   out_vertices[out_faces[idx]] = vertices[faces[idx]];
}

__global__ void smooth_normals(float3* __restrict__ vertices, int n_vertices,
   int* __restrict__ faces, int n_faces, float3* __restrict__ normals)
{  int i, block_id, idx;
   float3 e0, e1, n;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   if (idx >= n_vertices)
      return;

   n.x = n.y = n.z = 0;
   for (i=0; i < n_faces; i++) {
      if (faces[3*i] == idx || faces[3*i+1] == idx || faces[3*i+2] == idx) {
         e0 = vertices[faces[3*i]]   - vertices[faces[3*i+1]];
         e1 = vertices[faces[3*i+2]] - vertices[faces[3*i+1]];
         e0 = normalize(e0);
         e1 = normalize(e1);
         n  = n + cross(e0, e1);
      }
   }
   if (dot(n, n) < EPSILON)
      normals[idx].x = normals[idx].y = normals[idx].z = 0;
   else
      normals[idx] = normalize(n);
}

__global__ void print_threads()
{  int block_id, idx;

   block_id = blockIdx.y * gridDim.x + blockIdx.x;
   idx = block_id * blockDim.x + threadIdx.x;
   printf("Block: (%d, %d, %d), "
          "Thread: (%d, %d, %d), "
          "BDim: (%d,%d,%d), "
          "GDim: (%d,%d,%d), idx: %d\n",
          blockIdx.x, blockIdx.y, blockIdx.z,
          threadIdx.x, threadIdx.y, threadIdx.z,
          blockDim.x, blockDim.y, blockDim.z,
          gridDim.x, gridDim.y, gridDim.z, idx);
}
